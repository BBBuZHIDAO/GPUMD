#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan and GPUMD development team
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/

/*-----------------------------------------------------------------------------------------------100
Calculate:
    Angular-dependent Radial distribution function (AngularRDF)
--------------------------------------------------------------------------------------------------*/

#include "angular_rdf.cuh"
#include "force/neighbor.cuh"
#include "model/atom.cuh"
#include "model/box.cuh"
#include "model/group.cuh"
#include "parse_utilities.cuh"
#include "utilities/common.cuh"
#include "utilities/error.cuh"
#include "utilities/gpu_macro.cuh"
#include "utilities/read_file.cuh"
#include <cstring>

namespace
{
// GPU核函数：计算单一原子类型的RDF
static __global__ void gpu_find_rdf_ON1(
  //__global__,  CPU调用，GPU执行; __device__, 只能在GPU上调用和执行;
  //__host__, 只能在CPU上调用和执行.
  const int N,                            // 总原子数
  const double density,                   // 系统密度
  const Box box,                          // 模拟盒子
  const int* __restrict__ cell_counts,    // 每个晶胞的原子数
  const int* __restrict__ cell_count_sum, // 晶胞原子数累积和
  const int* __restrict__ cell_contents,  // 晶胞中的原子索引
  const int nx,
  const int ny,
  const int nz,                       // 晶胞网格维度
  const double rc_inv,                // 截断半径倒数
  const double* __restrict__ x,       // 原子x坐标
  const double* __restrict__ y,       // 原子y坐标
  const double* __restrict__ z,       // 原子z坐标
  const double* __restrict__ radial_, // 径向距离数组
  const double* __restrict__ theta_,  // 角度距离数组
  double* rdf_,                       // RDF结果数组
  const int rdf_bins_,                // bin数量
  const int rdf_theta_bins_,          // bin数量
  const double r_step_,               // 步长
  const double theta_step_)           // 步长
{
  // 获取当前线程处理的原子索引
  const int n1 = blockIdx.x * blockDim.x + threadIdx.x;
  double rdf_PI = 3.14159265358979323846;

  if (n1 < N) {
    // 获取当前原子的坐标
    const double x1 = x[n1];
    const double y1 = y[n1];
    const double z1 = z[n1];

    // 计算当前原子所在的晶胞ID
    int cell_id;
    int cell_id_x, cell_id_y, cell_id_z;
    find_cell_id(box, x1, y1, z1, rc_inv, nx, ny, nz, cell_id_x, cell_id_y, cell_id_z, cell_id);

    // 根据周期性边界条件设置搜索范围
    const int z_lim = box.pbc_z ? 2 : 0;
    const int y_lim = box.pbc_y ? 2 : 0;
    const int x_lim = box.pbc_x ? 2 : 0;

    // 遍历邻近晶胞
    for (int k = -z_lim; k <= z_lim; ++k) {
      for (int j = -y_lim; j <= y_lim; ++j) {
        for (int i = -x_lim; i <= x_lim; ++i) {
          // 计算邻近晶胞ID并处理周期性边界
          int neighbor_cell = cell_id + k * nx * ny + j * nx + i;
          // ... (周期性边界处理代码)
          if (cell_id_x + i < 0)
            neighbor_cell += nx;
          if (cell_id_x + i >= nx)
            neighbor_cell -= nx;
          if (cell_id_y + j < 0)
            neighbor_cell += ny * nx;
          if (cell_id_y + j >= ny)
            neighbor_cell -= ny * nx;
          if (cell_id_z + k < 0)
            neighbor_cell += nz * ny * nx;
          if (cell_id_z + k >= nz)
            neighbor_cell -= nz * ny * nx;

          // 遍历邻近晶胞中的原子
          const int num_atoms_neighbor_cell = cell_counts[neighbor_cell];
          const int num_atoms_previous_cells = cell_count_sum[neighbor_cell];

          for (int m = 0; m < num_atoms_neighbor_cell; ++m) {
            const int n2 = cell_contents[num_atoms_previous_cells + m];
            if (n2 >= 0 && n2 < N && n1 != n2) {
              // 计算原子对之间的距离
              double x12 = x[n2] - x1;
              double y12 = y[n2] - y1;
              double z12 = z[n2] - z1;
              apply_mic(box, x12, y12, z12); // 最小镜像约定
              const double d2 = x12 * x12 + y12 * y12 + z12 * z12;
              double theta = atan2(y12, x12);

              // 更新RDF直方图
              for (int w = 0; w < rdf_bins_; w++) {
                double r_low = radial_[w] - r_step_ / 2;
                double r_up = radial_[w] + r_step_ / 2;
                if (d2 > r_low * r_low && d2 <= r_up * r_up) {
                  printf(
                    "d2: %f, r_low: %f, r_up: %f, theta: %f\n", d2, r_low, r_up, theta); // DEBUG
                  printf("x[n2]: %f, y[n2]: %f, z[n2]: %f\n", x[n2], y[n2], z[n2]);      // DEBUG
                  printf("x[n1]: %f, y[n1]: %f, z[n1]: %f\n", x[n1], y[n1], z[n1]);      // DEBUG
                  printf("x12: %f, y12: %f, z12: %f\n", x12, y12, z12);                  // DEBUG
                  printf(
                    "d2: %f, r_low: %f, r_up: %f, theta: %f\n", d2, r_low, r_up, theta); // DEBUG
                  for (int t = 0; t < rdf_theta_bins_; t++) {
                    double theta_low = theta_[t] - theta_step_ / 2;
                    double theta_up = theta_[t] + theta_step_ / 2;
                    if (theta > theta_low && theta <= theta_up) {
                      printf(
                        "theta: %f, theta_low: %f, theta_up: %f\n",
                        theta,
                        theta_low,
                        theta_up); // DEBUG
                      // RDF归一化因子计算
                      double shell_volume =
                        4.0 / 3.0 * rdf_PI * (r_up * r_up * r_up - r_low * r_low * r_low);
                      double theta_area = (theta_up - theta_low) / (2 * rdf_PI);
                      double bin_volume = theta_area * shell_volume;
                      rdf_[n1 * rdf_bins_ * rdf_theta_bins_ + w * rdf_theta_bins_ + t] +=
                        1 / (N * density * bin_volume);
                    }
                  }
                }
              }
            }
          }
        }
      }
    }
  }
}

static __global__ void gpu_find_rdf_ON1(
  const int N,
  const double density1,
  const double density2,
  const double num_atom1_,
  const double num_atom2_,
  const double atom_id1_,
  const double atom_id2_,
  const Box box,
  const int* __restrict__ cell_counts,
  const int* __restrict__ cell_count_sum,
  const int* __restrict__ cell_contents,
  const int nx,
  const int ny,
  const int nz,
  const double rc_inv,
  const double* __restrict__ x,
  const double* __restrict__ y,
  const double* __restrict__ z,
  const int* __restrict__ type,
  const double* __restrict__ radial_,
  const double* __restrict__ theta_,
  double* rdf_,
  const int rdf_bins_,
  const int rdf_theta_bins_,
  const double r_step_,
  const double theta_step_)
{
  const int n1 = blockIdx.x * blockDim.x + threadIdx.x;
  double rdf_PI = 3.14159265358979323846;
  if (n1 < N && type[n1] == atom_id1_) {
    const double x1 = x[n1];
    const double y1 = y[n1];
    const double z1 = z[n1];
    int cell_id;
    int cell_id_x;
    int cell_id_y;
    int cell_id_z;
    find_cell_id(box, x1, y1, z1, rc_inv, nx, ny, nz, cell_id_x, cell_id_y, cell_id_z, cell_id);

    const int z_lim = box.pbc_z ? 2 : 0;
    const int y_lim = box.pbc_y ? 2 : 0;
    const int x_lim = box.pbc_x ? 2 : 0;

    // get radial descriptors
    for (int k = -z_lim; k <= z_lim; ++k) {
      for (int j = -y_lim; j <= y_lim; ++j) {
        for (int i = -x_lim; i <= x_lim; ++i) {
          int neighbor_cell = cell_id + k * nx * ny + j * nx + i;
          if (cell_id_x + i < 0)
            neighbor_cell += nx;
          if (cell_id_x + i >= nx)
            neighbor_cell -= nx;
          if (cell_id_y + j < 0)
            neighbor_cell += ny * nx;
          if (cell_id_y + j >= ny)
            neighbor_cell -= ny * nx;
          if (cell_id_z + k < 0)
            neighbor_cell += nz * ny * nx;
          if (cell_id_z + k >= nz)
            neighbor_cell -= nz * ny * nx;

          const int num_atoms_neighbor_cell = cell_counts[neighbor_cell];
          const int num_atoms_previous_cells = cell_count_sum[neighbor_cell];

          for (int m = 0; m < num_atoms_neighbor_cell; ++m) {
            const int n2 = cell_contents[num_atoms_previous_cells + m];
            if (n2 >= 0 && n2 < N && n1 != n2 && type[n2] == atom_id2_) {
              double x12 = x[n2] - x1;
              double y12 = y[n2] - y1;
              double z12 = z[n2] - z1;
              apply_mic(box, x12, y12, z12);
              const double d2 = x12 * x12 + y12 * y12 + z12 * z12;
              double theta = atan2(y12, x12);
              for (int w = 0; w < rdf_bins_; w++) {
                double r_low = radial_[w] - r_step_ / 2;
                double r_up = radial_[w] + r_step_ / 2;
                if (d2 > r_low * r_low && d2 <= r_up * r_up) {
                  for (int t = 0; t < rdf_theta_bins_; t++) {
                    double theta_low = theta_[t] - theta_step_ / 2;
                    double theta_up = theta_[t] + theta_step_ / 2;
                    if (theta > theta_low && theta <= theta_up) {
                      printf("theta_low: %f, theta_up: %f\n", theta_low, theta_up); // DEBUG
                      double shell_volume =
                        4.0 / 3.0 * rdf_PI * (r_up * r_up * r_up - r_low * r_low * r_low);
                      double theta_area = (theta_up - theta_low) / (2 * rdf_PI);
                      double bin_volume = theta_area * shell_volume;
                      rdf_[n1 * rdf_bins_ * rdf_theta_bins_ + w * rdf_theta_bins_ + t] +=
                        1 / (num_atom1_ * density2 * bin_volume);
                      printf("shell_volume: %f\n", shell_volume); // DEBUG
                      printf("theta_area: %f\n", theta_area);     // DEBUG
                      printf("density2: %f\n", density2);         // DEBUG
                      printf("bin_volume: %f\n", bin_volume);     // DEBUG
                      printf("n1: %d, w: %d, t: %d\n", n1, w, t); // DEBUG
                      printf(
                        "rdf_[n1 * rdf_bins_ * rdf_theta_bins_ + w * rdf_theta_bins_ + t]: %f\n",
                        rdf_[n1 * rdf_bins_ * rdf_theta_bins_ + w * rdf_theta_bins_ + t]); // DEBUG
                      printf(
                        "n1 * rdf_bins_ * rdf_theta_bins_ + w * rdf_theta_bins_ + t  : %d\n",
                        n1 * rdf_bins_ * rdf_theta_bins_ + w * rdf_theta_bins_ + t); // DEBUG
                    }
                  }
                }
              }
            }
          }
        }
      }
    }
  } else if (n1 < N && type[n1] == atom_id2_) {
    const double x1 = x[n1];
    const double y1 = y[n1];
    const double z1 = z[n1];
    int cell_id;
    int cell_id_x;
    int cell_id_y;
    int cell_id_z;
    find_cell_id(box, x1, y1, z1, rc_inv, nx, ny, nz, cell_id_x, cell_id_y, cell_id_z, cell_id);

    const int z_lim = box.pbc_z ? 2 : 0;
    const int y_lim = box.pbc_y ? 2 : 0;
    const int x_lim = box.pbc_x ? 2 : 0;

    // get radial descriptors
    for (int k = -z_lim; k <= z_lim; ++k) {
      for (int j = -y_lim; j <= y_lim; ++j) {
        for (int i = -x_lim; i <= x_lim; ++i) {
          int neighbor_cell = cell_id + k * nx * ny + j * nx + i;
          if (cell_id_x + i < 0)
            neighbor_cell += nx;
          if (cell_id_x + i >= nx)
            neighbor_cell -= nx;
          if (cell_id_y + j < 0)
            neighbor_cell += ny * nx;
          if (cell_id_y + j >= ny)
            neighbor_cell -= ny * nx;
          if (cell_id_z + k < 0)
            neighbor_cell += nz * ny * nx;
          if (cell_id_z + k >= nz)
            neighbor_cell -= nz * ny * nx;

          const int num_atoms_neighbor_cell = cell_counts[neighbor_cell];
          const int num_atoms_previous_cells = cell_count_sum[neighbor_cell];

          for (int m = 0; m < num_atoms_neighbor_cell; ++m) {
            const int n2 = cell_contents[num_atoms_previous_cells + m];
            if (n2 >= 0 && n2 < N && n1 != n2 && type[n2] == atom_id1_) {
              double x12 = x1 - x[n2];
              double y12 = y1 - y[n2];
              double z12 = z1 - z[n2];
              apply_mic(box, x12, y12, z12);
              const double d2 = x12 * x12 + y12 * y12 + z12 * z12;
              double theta = atan2(y12, x12);
              for (int w = 0; w < rdf_bins_; w++) {
                double r_low = radial_[w] - r_step_ / 2;
                double r_up = radial_[w] + r_step_ / 2;
                if (d2 > r_low * r_low && d2 <= r_up * r_up) {
                  for (int t = 0; t < rdf_theta_bins_; t++) {
                    double theta_low = theta_[t] - theta_step_ / 2;
                    double theta_up = theta_[t] + theta_step_ / 2;
                    if (theta > theta_low && theta <= theta_up) {
                      double shell_volume =
                        4.0 / 3.0 * rdf_PI * (r_up * r_up * r_up - r_low * r_low * r_low);
                      double theta_area = (theta_up - theta_low) / (2 * rdf_PI);
                      double bin_volume = theta_area * shell_volume;
                      rdf_[n1 * rdf_bins_ * rdf_theta_bins_ + w * rdf_theta_bins_ + t] +=
                        1 / (num_atom2_ * density1 * bin_volume);
                      printf("shell_volume: %f\n", shell_volume); // DEBUG
                      printf("theta_area: %f\n", theta_area);     // DEBUG
                      printf("density2: %f\n", density2);         // DEBUG
                      printf("bin_volume: %f\n", bin_volume);     // DEBUG
                      printf("n1: %d, w: %d, t: %d\n", n1, w, t); // DEBUG
                      printf(
                        "rdf_[n1 * rdf_bins_ * rdf_theta_bins_ + w * rdf_theta_bins_ + t]: %f\n",
                        rdf_[n1 * rdf_bins_ * rdf_theta_bins_ + w * rdf_theta_bins_ + t]); // DEBUG
                      printf(
                        "n1 * rdf_bins_ * rdf_theta_bins_ + w * rdf_theta_bins_ + t  : %d\n",
                        n1 * rdf_bins_ * rdf_theta_bins_ + w * rdf_theta_bins_ + t); // DEBUG
                    }
                  }
                }
              }
            }
          }
        }
      }
    }
  }
}
} // namespace

// Function to calculate angular radial distribution function (RDF)
// Parameters:
// bead - current bead index
// rdf_atom_count - total number of atom types for RDF calculation
// rdf_atom_ - current atom type index
// atom_id1_ - array of first atom type IDs
// atom_id2_ - array of second atom type IDs
// atom_id1_typesize - number of atoms of first type for each pair
// atom_id2_typesize - number of atoms of second type for each pair
// density1 - number density of first atom type
// density2 - number density of second atom type
// rc - cutoff radius
// box - simulation box
// type - atom type array
// position_per_atom - atom positions array
// cell_count - number of atoms in each cell
// cell_count_sum - cumulative sum of atoms in cells
// cell_contents - atom indices in each cell
// num_bins_0,1,2 - number of cells in x,y,z directions
// rc_inv_cell_list - inverse of cell list cutoff
// radial_ - radial distance array
// rdf_g_ - RDF array
// rdf_r_bins_ - number of RDF histogram bins
// r_step_ - RDF bin width
void AngularRDF::find_angular_rdf(
  const int bead,
  const int rdf_atom_count,
  const int rdf_atom_, // 当前原子对的编号
  int* atom_id1_,
  int* atom_id2_,
  std::vector<int>& atom_id1_typesize,
  std::vector<int>& atom_id2_typesize,
  std::vector<double>& density1,
  std::vector<double>& density2,
  double rc,
  Box& box,
  const GPU_Vector<int>& type,
  const GPU_Vector<double>& position_per_atom,
  GPU_Vector<int>& cell_count,
  GPU_Vector<int>& cell_count_sum,
  GPU_Vector<int>& cell_contents,
  int num_bins_0,
  int num_bins_1,
  int num_bins_2,
  const double rc_inv_cell_list,
  GPU_Vector<double>& radial_, // num_atoms_ * rdf_r_bins_ * rdf_theta_bins_,
  GPU_Vector<double>& theta_,
  GPU_Vector<double>& rdf_g_,
  const int rdf_r_bins_,
  const int rdf_theta_bins_,
  const double r_step_,
  const double theta_step_)
{
  const int N = position_per_atom.size() / 3;
  const int block_size = 256;
  const int grid_size = (N - 1) / block_size + 1;
  const double* x = position_per_atom.data();
  const double* y = position_per_atom.data() + N;
  const double* z = position_per_atom.data() + N * 2;

  double* rdf_g_ind = rdf_g_.data() + rdf_atom_ * N * rdf_r_bins_ * rdf_theta_bins_;

  if (rdf_atom_ == 0) {
    gpu_find_rdf_ON1<<<grid_size, block_size>>>(
      N,
      density1[rdf_atom_],
      box,
      cell_count.data(),
      cell_count_sum.data(),
      cell_contents.data(),
      num_bins_0,
      num_bins_1,
      num_bins_2,
      rc_inv_cell_list,
      x,
      y,
      z,
      radial_.data(),
      theta_.data(),
      rdf_g_ind,
      rdf_r_bins_,
      rdf_theta_bins_,
      r_step_,
      theta_step_);
    GPU_CHECK_KERNEL
    // GPU_CHECK_KERNEL 是一个用于检查CUDA核函数执行是否成功的宏定义。

  } else {
    gpu_find_rdf_ON1<<<grid_size, block_size>>>(
      N,
      density1[rdf_atom_],
      density2[rdf_atom_],
      atom_id1_typesize[rdf_atom_ - 1],
      atom_id2_typesize[rdf_atom_ - 1],
      atom_id1_[rdf_atom_ - 1],
      atom_id2_[rdf_atom_ - 1],
      box,
      cell_count.data(),
      cell_count_sum.data(),
      cell_contents.data(),
      num_bins_0,
      num_bins_1,
      num_bins_2,
      rc_inv_cell_list,
      x,
      y,
      z,
      type.data(),
      radial_.data(),
      theta_.data(),
      rdf_g_ind,
      rdf_r_bins_,
      rdf_theta_bins_,
      r_step_,
      theta_step_);
    GPU_CHECK_KERNEL
  }
}

// 预处理函数:初始化角度相关RDF计算所需的数据结构
void AngularRDF::preprocess(
  const bool is_pimd,              // 是否为路径积分分子动力学
  const int number_of_beads,       // PIMD珠子数量
  const int num_atoms,             // 系统总原子数
  std::vector<int>& cpu_type_size) // 每种原子类型的数量
{
  // 如果不需要计算RDF则直接返回
  if (!compute_)
    return;

  // 如果为PIMD则直接返回, 当前不支持PIMD
  if (is_pimd) {
    return;
  }

  // 计算径向步长
  r_step_ = r_cut_ / rdf_r_bins_;

  // 计算角度步长
  theta_step_ = 2 * M_PI / rdf_theta_bins_; // 总角度为360度，角度步长为360度除以角度bin数量

  // 初始化径向距离数组
  std::vector<double> radial_cpu(rdf_r_bins_);
  for (int i = 0; i < rdf_r_bins_; i++) {
    radial_cpu[i] = i * r_step_ + r_step_ / 2; // 每个bin的中心位置
  }
  radial_.resize(rdf_r_bins_);
  radial_.copy_from_host(radial_cpu.data()); // 将数据复制到GPU

  // 初始化角度距离数组
  std::vector<double> theta_cpu(rdf_theta_bins_);
  for (int i = 0; i < rdf_theta_bins_; i++) {
    theta_cpu[i] =
      -M_PI + i * theta_step_ + theta_step_ / 2; // 每个bin的中心位置, atan2 返回值范围为-pi到pi
  }
  theta_.resize(rdf_theta_bins_);
  theta_.copy_from_host(theta_cpu.data()); // 将数据复制到GPU

  // 设置原子数相关参数
  rdf_N_ = num_atoms;
  num_atoms_ =
    num_atoms * rdf_atom_count; // rdf_atom_count为原子对数量, num_atoms_为考虑所有原子对后的原子数

  // 分配密度数组空间
  density1.resize(rdf_atom_count);
  density2.resize(rdf_atom_count);

  // 分配原子类型大小数组空间
  atom_id1_typesize.resize(
    rdf_atom_count - 1); // 我们总是考虑计算所有原子间的Angular RDF，这里只存储Partial AngularRDF
  atom_id2_typesize.resize(rdf_atom_count - 1);

  // 初始化原子类型数目数组
  for (int a = 0; a < rdf_atom_count - 1; a++) {
    atom_id1_typesize[a] = cpu_type_size[atom_id1_[a]];
    atom_id2_typesize[a] = cpu_type_size[atom_id2_[a]];
  }

  rdf_g_.resize(
    num_atoms_ * rdf_r_bins_ * rdf_theta_bins_,
    0); // 注意这里实际是一个三维数组，rdf_g_的维度为num_atoms_ * rdf_r_bins_
  rdf_.resize(num_atoms_ * rdf_r_bins_ * rdf_theta_bins_, 0);
  cell_count.resize(num_atoms);
  cell_count_sum.resize(num_atoms);
  cell_contents.resize(num_atoms);
}

void AngularRDF::process(
  const bool is_pimd, const int number_of_steps, const int step, Box& box, Atom& atom)
{
  // 如果为PIMD则直接返回, 当前不支持PIMD
  if (is_pimd) {
    return;
  }

  // 如果不需要计算RDF则直接返回
  if (!compute_)
    return;

  // 如果步数不是采样间隔的倍数则直接返回
  if ((step + 1) % num_interval_ != 0) {
    return;
  }

  // 重复次数
  num_repeat_++;

  // 计算数密度
  density1[0] = rdf_N_ / box.get_volume();
  density2[0] = rdf_N_ / box.get_volume();
  for (int a = 0; a < rdf_atom_count - 1; a++) {
    density1[a + 1] = atom_id1_typesize[a] / box.get_volume();
    density2[a + 1] = atom_id2_typesize[a] / box.get_volume();
  }

  int classical = 0;
  const double rc_cell_list = 0.5 * r_cut_;
  const double rc_inv_cell_list = 2.0 / r_cut_;
  int num_bins[3];
  box.get_num_bins(rc_cell_list, num_bins);
  find_cell_list(
    rc_cell_list, num_bins, box, atom.position_per_atom, cell_count, cell_count_sum, cell_contents);

  for (int a = 0; a < rdf_atom_count; a++) {
    find_angular_rdf(
      classical,
      rdf_atom_count,
      a,
      atom_id1_,
      atom_id2_,
      atom_id1_typesize,
      atom_id2_typesize,
      density1,
      density2,
      r_cut_,
      box,
      atom.type,
      atom.position_per_atom,
      cell_count,
      cell_count_sum,
      cell_contents,
      num_bins[0],
      num_bins[1],
      num_bins[2],
      rc_inv_cell_list,
      radial_,
      theta_,
      rdf_g_,
      rdf_r_bins_,
      rdf_theta_bins_,
      r_step_,
      theta_step_);
  }
}

void AngularRDF::postprocess(const bool is_pimd, const int number_of_beads)
{
  if (!compute_)
    return;
  if (is_pimd)
    return;

  CHECK(gpuMemcpy(
    rdf_.data(),
    rdf_g_.data(),
    sizeof(double) * num_atoms_ * rdf_r_bins_ * rdf_theta_bins_,
    gpuMemcpyDeviceToHost));
  CHECK(gpuDeviceSynchronize()); // needed for pre-Pascal GPU

  printf(
    "rdf_[2,0,33,74]1: %f\n",
    rdf_
      [2 * rdf_N_ * rdf_r_bins_ * rdf_theta_bins_ + 0 * rdf_r_bins_ * rdf_theta_bins_ +
       33 * rdf_theta_bins_ + 74]);
  printf(
    "rdf_[3,1,33,24]2: %f\n",
    rdf_
      [3 * rdf_N_ * rdf_r_bins_ * rdf_theta_bins_ + 1 * rdf_r_bins_ * rdf_theta_bins_ +
       33 * rdf_theta_bins_ + 24]);
  printf(
    "rdf_[2,1,33,74]1: %f\n",
    rdf_
      [2 * rdf_N_ * rdf_r_bins_ * rdf_theta_bins_ + 1 * rdf_r_bins_ * rdf_theta_bins_ +
       33 * rdf_theta_bins_ + 74]);
  printf(
    "rdf_[3,0,33,24]2: %f\n",
    rdf_
      [3 * rdf_N_ * rdf_r_bins_ * rdf_theta_bins_ + 0 * rdf_r_bins_ * rdf_theta_bins_ +
       33 * rdf_theta_bins_ + 24]);

  std::vector<double> rdf_average(rdf_atom_count * rdf_r_bins_ * rdf_theta_bins_, 0.0);
  for (int a = 0; a < rdf_atom_count; a++) {
    for (int m = 0; m < rdf_N_; m++) {
      for (int x = 0; x < rdf_r_bins_; x++) {
        for (int t = 0; t < rdf_theta_bins_; t++) {
          rdf_average[a * rdf_r_bins_ * rdf_theta_bins_ + x * rdf_theta_bins_ + t] +=
            rdf_
              [a * rdf_N_ * rdf_r_bins_ * rdf_theta_bins_ + m * rdf_r_bins_ * rdf_theta_bins_ +
               x * rdf_theta_bins_ + t] /
            num_repeat_;
        }
      }
    }
  }
  printf(
    "rdf_average[special]1: %f\n",
    rdf_average[2 * rdf_r_bins_ * rdf_theta_bins_ + 33 * rdf_theta_bins_ + 74]);
  printf(
    "rdf_average[special]2: %f\n",
    rdf_average[3 * rdf_r_bins_ * rdf_theta_bins_ + 33 * rdf_theta_bins_ + 24]);
  FILE* fid = fopen("angular_rdf.out", "a");
  fprintf(fid, "#radius theta");
  // print the header
  for (int a = 0; a < rdf_atom_count; a++) {
    if (a == 0) {
      fprintf(fid, " total");
    } else {
      fprintf(fid, " type_%d_%d", atom_id1_[a - 1], atom_id2_[a - 1]);
    }
  }
  fprintf(fid, "\n");
  // print the data
  for (int nc = 0; nc < rdf_r_bins_; nc++) {
    for (int tc = 0; tc < rdf_theta_bins_; tc++) {
      fprintf(
        fid, "%.5f %.5f", nc * r_step_ + r_step_ / 2, -M_PI + tc * theta_step_ + theta_step_ / 2);
      for (int a = 0; a < rdf_atom_count; a++) {
        if (a == 0) {
          fprintf(fid, " %.5f", rdf_average[nc * rdf_theta_bins_ + tc]);
        } else {
          fprintf(
            fid,
            " %.5f",
            (atom_id1_[a - 1] == atom_id2_[a - 1])
              ? rdf_average[a * rdf_r_bins_ * rdf_theta_bins_ + nc * rdf_theta_bins_ + tc]
              : rdf_average[a * rdf_r_bins_ * rdf_theta_bins_ + nc * rdf_theta_bins_ + tc] / 2);
        }
      }
      fprintf(fid, "\n");
    }
  }
  fflush(fid);
  fclose(fid);

  compute_ = false;
  for (int s = 0; s < 6; s++) {
    atom_id1_[s] = -1;
    atom_id2_[s] = -1;
  }
  rdf_atom_count = 1;
  num_repeat_ = 0;
}

void AngularRDF::parse(
  const char** param,
  const int num_param,
  Box& box,
  const int number_of_types,
  const int number_of_steps)
{
  printf("Compute Angular RDF.\n");
  compute_ = true;

  if (num_param < 5) {
    PRINT_INPUT_ERROR("compute_angular_rdf should have at least 4 parameters.\n");
  }
  if (num_param > 23) {
    PRINT_INPUT_ERROR("compute_angular_rdf has too many parameters.\n");
  }

  // radial cutoff
  if (!is_valid_real(param[1], &r_cut_)) {
    PRINT_INPUT_ERROR("radial cutoff should be a number.\n");
  }
  if (r_cut_ <= 0) {
    PRINT_INPUT_ERROR("radial cutoff should be positive.\n");
  }
  double thickness_half[3] = {
    box.get_volume() / box.get_area(0) / 2.5,
    box.get_volume() / box.get_area(1) / 2.5,
    box.get_volume() / box.get_area(2) / 2.5};
  if (r_cut_ > thickness_half[0] || r_cut_ > thickness_half[1] || r_cut_ > thickness_half[2]) {
    std::string message =
      "The box has a thickness < 2.5 RDF radial cutoffs in a periodic direction.\n"
      "                Please increase the periodic direction(s).\n";
    PRINT_INPUT_ERROR(message.c_str());
  }
  printf("    radial cutoff %g.\n", r_cut_);

  // number of bins
  if (!is_valid_int(param[2], &rdf_r_bins_)) {
    PRINT_INPUT_ERROR("number of bins should be an integer.\n");
  }
  if (rdf_r_bins_ <= 20) {
    PRINT_INPUT_ERROR("A larger nbins is recommended.\n");
  }

  if (rdf_r_bins_ > 500) {
    PRINT_INPUT_ERROR("A smaller nbins is recommended.\n");
  }

  printf("    radial cutoff will be divided into %d bins.\n", rdf_r_bins_);

  // 角度方向的bin数量
  if (!is_valid_int(param[3], &rdf_theta_bins_)) {
    PRINT_INPUT_ERROR("number of theta bins should be an integer.\n");
  }
  if (rdf_theta_bins_ <= 20) {
    PRINT_INPUT_ERROR("A larger ntheta is recommended.\n");
  }
  printf("    theta cutoff will be divided into %d bins.\n", rdf_theta_bins_);

  // sample interval
  if (!is_valid_int(param[4], &num_interval_)) {
    PRINT_INPUT_ERROR("interval step per sample should be an integer.\n");
  }
  if (num_interval_ <= 0) {
    PRINT_INPUT_ERROR("interval step per sample should be positive.\n");
  }
  printf("    Angular RDF sample interval is %d step.\n", num_interval_);

  // Process optional arguments
  for (int k = 5; k < num_param; k += 3) {
    if (strcmp(param[k], "atom") == 0) {
      int k_a = (k - 5) / 3;
      rdf_atom_count++;
      if (!is_valid_int(param[k + 1], &atom_id1_[k_a])) {
        PRINT_INPUT_ERROR("atom type index1 should be an integer.\n");
      }
      if (atom_id1_[k_a] < 0) {
        PRINT_INPUT_ERROR("atom type index1 should be non-negative.\n");
      }
      if (atom_id1_[k_a] > number_of_types) {
        PRINT_INPUT_ERROR("atom type index1 should be less than number of atomic types.\n");
      }
      if (!is_valid_int(param[k + 2], &atom_id2_[k_a])) {
        PRINT_INPUT_ERROR("atom type index2 should be an integer.\n");
      }
      if (atom_id2_[k_a] < 0) {
        PRINT_INPUT_ERROR("atom type index2 should be non-negative.\n");
      }
      if (atom_id2_[k_a] > number_of_types) {
        PRINT_INPUT_ERROR("atom type index1 should be less than number of atomic types.\n");
      }
    } else {
      PRINT_INPUT_ERROR("Unrecognized argument in compute_angular_rdf.\n");
    }
  }
}
