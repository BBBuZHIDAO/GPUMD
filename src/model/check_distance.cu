#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan and GPUMD development team
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/

/*----------------------------------------------------------------------------80
Calculate the distance between any two atoms in the model.xyz file.
------------------------------------------------------------------------------*/

#include "atom.cuh"
#include "box.cuh"
#include "check_distance.cuh"
#include "utilities/error.cuh"
#include <cmath>

void applyMicOne(double& x12)
{
  if (x12 < -0.5) {
    x12 += 1.0;
  } else if (x12 > +0.5) {
    x12 -= 1.0;
  }
}

void applyMic(const Box& box, double& x12, double& y12, double& z12)
{
  int pbc[3] = {box.pbc_x, box.pbc_y, box.pbc_z};
  double s[3];
  for (int i = 0; i < 3; ++i) {
    s[i] = box.cpu_h[9 + i * 3] * x12 + box.cpu_h[10 + i * 3] * y12 + box.cpu_h[11 + i * 3] * z12;
    if (pbc[i])
      applyMicOne(s[i]);
  }
  x12 = box.cpu_h[0] * s[0] + box.cpu_h[3] * s[1] + box.cpu_h[6] * s[2];
  y12 = box.cpu_h[1] * s[0] + box.cpu_h[4] * s[1] + box.cpu_h[7] * s[2];
  z12 = box.cpu_h[2] * s[0] + box.cpu_h[5] * s[1] + box.cpu_h[8] * s[2];
}

void findCell(
  const Box& box, const double* thickness, const double* r, const int* numCells, int* cell)
{
  double s[3];
  for (int d = 0; d < 3; ++d) {
    s[d] =
      box.cpu_h[9 + d * 3] * r[0] + box.cpu_h[10 + d * 3] * r[1] + box.cpu_h[11 + d * 3] * r[2];
    cell[d] = floor(s[d] * thickness[d] * 0.2);
    if (cell[d] < 0)
      cell[d] += numCells[d];
    if (cell[d] >= numCells[d])
      cell[d] -= numCells[d];
  }
  cell[3] = cell[0] + numCells[0] * (cell[1] + numCells[1] * cell[2]);
}

void calculate_min_atomic_distance(const Atom& atom, const Box& box)
{
  const int N = atom.number_of_atoms;
  const double* pos = atom.cpu_position_per_atom.data();

  double min_distance = 5.0;
  int min_n1 = -1, min_n2 = -1;

  int cell[4], numCells[4];
  double thickness[3];
  for (int i = 0; i < 3; ++i) {
    thickness[i] = sqrt(
      box.cpu_h[i] * box.cpu_h[i] + box.cpu_h[i + 3] * box.cpu_h[i + 3] +
      box.cpu_h[i + 6] * box.cpu_h[i + 6]);
    numCells[i] = std::max(1, static_cast<int>(ceil(thickness[i] * 0.2)));
  }
  numCells[3] = numCells[0] * numCells[1] * numCells[2];

  std::vector<int> cellContents(N, 0);
  std::vector<int> cellCount(numCells[3], 0);
  std::vector<int> cellCountSum(numCells[3], 0);
  std::fill(cellCount.begin(), cellCount.end(), 0);

  for (int n = 0; n < N; ++n) {
    const double r[3] = {pos[n], pos[n + N], pos[n + 2 * N]};
    findCell(box, thickness, r, numCells, cell);
    ++cellCount[cell[3]];
  }

  for (int i = 1; i < numCells[3]; ++i) {
    cellCountSum[i] = cellCountSum[i - 1] + cellCount[i - 1];
  }

  for (int n = 0; n < N; ++n) {
    const double r[3] = {pos[n], pos[n + N], pos[n + 2 * N]};
    findCell(box, thickness, r, numCells, cell);
    cellContents[cellCountSum[cell[3]] + cellCount[cell[3]]] = n;
    ++cellCount[cell[3]];
  }

  for (int n1 = 0; n1 < N; ++n1) {
    const double r1[3] = {pos[n1], pos[n1 + N], pos[n1 + 2 * N]};
    findCell(box, thickness, r1, numCells, cell);
    for (int k = -1; k <= 1; ++k) {
      for (int j = -1; j <= 1; ++j) {
        for (int i = -1; i <= 1; ++i) {
          int neighborCell = cell[3] + (k * numCells[1] + j) * numCells[0] + i;
          if (cell[0] + i < 0)
            neighborCell += numCells[0];
          if (cell[0] + i >= numCells[0])
            neighborCell -= numCells[0];
          if (cell[1] + j < 0)
            neighborCell += numCells[1] * numCells[0];
          if (cell[1] + j >= numCells[1])
            neighborCell -= numCells[1] * numCells[0];
          if (cell[2] + k < 0)
            neighborCell += numCells[3];
          if (cell[2] + k >= numCells[2])
            neighborCell -= numCells[3];
          for (int m = 0; m < cellCount[neighborCell]; ++m) {
            const int n2 = cellContents[cellCountSum[neighborCell] + m];
            if (n1 < n2) {
              double x12 = pos[n2] - r1[0];
              double y12 = pos[n2 + N] - r1[1];
              double z12 = pos[n2 + 2 * N] - r1[2];
              applyMic(box, x12, y12, z12);
              if (fabs(x12) > 2.0 || fabs(y12) > 2.0 || fabs(z12) > 2.0)
                continue;
              const double d2 = x12 * x12 + y12 * y12 + z12 * z12;
              if (d2 >= 4.0)
                continue;

              double distance = d2;
              if (distance < min_distance) {
                min_distance = distance;
                min_n1 = n1;
                min_n2 = n2;
              }
            }
          }
        }
      }
    }
  }
  double mini_distance = sqrt(min_distance);

  if (mini_distance < 1.0) {
    printf(
      "Error: Minimum distance (%f Å) between atoms %d (%s) and %d (%s) is less than 1 Å.\n",
      mini_distance,
      min_n1,
      atom.cpu_atom_symbol[min_n1].c_str(),
      min_n2,
      atom.cpu_atom_symbol[min_n2].c_str());
    PRINT_INPUT_ERROR("There are two atoms with a distance less than 1 Å.");
  } else if (min_n1 != -1 && min_n2 != -1) {
    printf(
      "Minimum distance between atoms %d (%s) and %d (%s): %f Å\n",
      min_n1,
      atom.cpu_atom_symbol[min_n1].c_str(),
      min_n2,
      atom.cpu_atom_symbol[min_n2].c_str(),
      mini_distance);
  }
}
