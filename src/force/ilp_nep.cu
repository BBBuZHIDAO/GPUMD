#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan and GPUMD development team
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/

/*----------------------------------------------------------------------------80
The class dealing with the interlayer potential(ILP) and neuroevolution 
potential(NEP).
TODO:
------------------------------------------------------------------------------*/

#include "ilp_nep.cuh"
#include "neighbor.cuh"
#include "utilities/error.cuh"
#include "utilities/common.cuh"
#include "utilities/gpu_macro.cuh"


const std::string ELEMENTS[NUM_ELEMENTS] = {
  "H",  "He", "Li", "Be", "B",  "C",  "N",  "O",  "F",  "Ne", "Na", "Mg", "Al", "Si", "P",  "S",
  "Cl", "Ar", "K",  "Ca", "Sc", "Ti", "V",  "Cr", "Mn", "Fe", "Co", "Ni", "Cu", "Zn", "Ga", "Ge",
  "As", "Se", "Br", "Kr", "Rb", "Sr", "Y",  "Zr", "Nb", "Mo", "Tc", "Ru", "Rh", "Pd", "Ag", "Cd",
  "In", "Sn", "Sb", "Te", "I",  "Xe", "Cs", "Ba", "La", "Ce", "Pr", "Nd", "Pm", "Sm", "Eu", "Gd",
  "Tb", "Dy", "Ho", "Er", "Tm", "Yb", "Lu", "Hf", "Ta", "W",  "Re", "Os", "Ir", "Pt", "Au", "Hg",
  "Tl", "Pb", "Bi", "Po", "At", "Rn", "Fr", "Ra", "Ac", "Th", "Pa", "U",  "Np", "Pu"};


ILP_NEP::ILP_NEP(FILE* fid_ilp, FILE* fid_nep_map, int num_types, int num_atoms)
{
  // read ILP elements
  printf("Use %d-element ILP potential with elements:\n", num_types);
  if (!(num_types >= 1 && num_types <= MAX_TYPE_ILP_NEP)) {
    PRINT_INPUT_ERROR("Incorrect type number of ILP parameters.\n");
  }
  for (int n = 0; n < num_types; ++n) {
    char atom_symbol[10];
    int count = fscanf(fid_ilp, "%s", atom_symbol);
    PRINT_SCANF_ERROR(count, 1, "Reading error for ILP potential.");
    printf(" %s", atom_symbol);
  }
  printf("\n");

  // read ILP group method
  PRINT_SCANF_ERROR(fscanf(fid_ilp, "%d", &ilp_group_method), 1, 
  "Reading error for ILP group method.");
  printf("Use group method %d to identify molecule for ILP.\n", ilp_group_method);

  // read ILP parameters
  float beta, alpha, delta, epsilon, C, d, sR;
  float reff, C6, S, rcut_ilp, rcut_global;
  rc = 0.0;
  for (int n = 0; n < num_types; ++n) {
    for (int m = 0; m < num_types; ++m) {
      int count = fscanf(fid_ilp, "%f%f%f%f%f%f%f%f%f%f%f%f", \
      &beta, &alpha, &delta, &epsilon, &C, &d, &sR, &reff, &C6, &S, \
      &rcut_ilp, &rcut_global);
      PRINT_SCANF_ERROR(count, 12, "Reading error for ILP potential.");

      ilp_para.C[n][m] = C;
      ilp_para.C_6[n][m] = C6;
      ilp_para.d[n][m] = d;
      ilp_para.d_Seff[n][m] = d / sR / reff;
      ilp_para.epsilon[n][m] = epsilon;
      ilp_para.z0[n][m] = beta;
      ilp_para.lambda[n][m] = alpha / beta;
      ilp_para.delta2inv[n][m] = 1.0 / (delta * delta);
      ilp_para.S[n][m] = S;
      ilp_para.rcutsq_ilp[n][m] = rcut_ilp * rcut_ilp;
      ilp_para.rcut_global[n][m] = rcut_global;
      float meV = 1e-3 * S;
      ilp_para.C[n][m] *= meV;
      ilp_para.C_6[n][m] *= meV;
      ilp_para.epsilon[n][m] *= meV;

      if (rc < rcut_global)
        rc = rcut_global;
    }
  }

  // read NEP group method from nep map file
  PRINT_SCANF_ERROR(fscanf(fid_nep_map, "%d", &nep_group_method), 1, 
  "Reading error for NEP group method.");
  printf("Use group method %d to identify molecule for NEP.\n", nep_group_method);

  // read the number of NEP file
  PRINT_SCANF_ERROR(fscanf(fid_nep_map, "%d", &num_nep), 1, 
  "Reading error for the number of NEP file.");
  printf("NEP file number: %d\n", num_nep);

  
  // read NEP parameter

  // initialize neighbor lists and some temp vectors
  int max_neighbor_number = min(num_atoms, CUDA_MAX_NL_ILP_NEP_CBN);
  ilp_data.NN.resize(num_atoms);
  ilp_data.NL.resize(num_atoms * max_neighbor_number);
  ilp_data.cell_count.resize(num_atoms);
  ilp_data.cell_count_sum.resize(num_atoms);
  ilp_data.cell_contents.resize(num_atoms);

  // init ilp neighbor list
  ilp_data.ilp_NN.resize(num_atoms);
  ilp_data.ilp_NL.resize(num_atoms * MAX_ILP_NEIGHBOR_CBN);
  ilp_data.reduce_NL.resize(num_atoms * max_neighbor_number);
  ilp_data.big_ilp_NN.resize(num_atoms);
  ilp_data.big_ilp_NL.resize(num_atoms * MAX_BIG_ILP_NEIGHBOR_CBN);

  ilp_data.f12x.resize(num_atoms * max_neighbor_number);
  ilp_data.f12y.resize(num_atoms * max_neighbor_number);
  ilp_data.f12z.resize(num_atoms * max_neighbor_number);

  ilp_data.f12x_ilp_neigh.resize(num_atoms * MAX_ILP_NEIGHBOR_CBN);
  ilp_data.f12y_ilp_neigh.resize(num_atoms * MAX_ILP_NEIGHBOR_CBN);
  ilp_data.f12z_ilp_neigh.resize(num_atoms * MAX_ILP_NEIGHBOR_CBN);

  // init constant cutoff coeff
  float h_tap_coeff[8] = \
    {1.0f, 0.0f, 0.0f, 0.0f, -35.0f, 84.0f, -70.0f, 20.0f};
  CHECK(gpuMemcpyToSymbol(Tap_coeff_tmd, h_tap_coeff, 8 * sizeof(float)));

  // set ilp_flag to 1
  ilp_flag = 1;
}