#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan and GPUMD development team
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/

/*----------------------------------------------------------------------------80
The class dealing with the interlayer potential(ILP) and neuroevolution 
potential(NEP).
TODO:
------------------------------------------------------------------------------*/

#include "ilp_nep.cuh"
#include "neighbor.cuh"
#include "utilities/error.cuh"
#include "utilities/common.cuh"
#include "utilities/gpu_macro.cuh"
#include "utilities/nep_utilities.cuh"
#include <iostream>
#include <fstream>
#include <string>


const std::string ELEMENTS[NUM_ELEMENTS] = {
  "H",  "He", "Li", "Be", "B",  "C",  "N",  "O",  "F",  "Ne", "Na", "Mg", "Al", "Si", "P",  "S",
  "Cl", "Ar", "K",  "Ca", "Sc", "Ti", "V",  "Cr", "Mn", "Fe", "Co", "Ni", "Cu", "Zn", "Ga", "Ge",
  "As", "Se", "Br", "Kr", "Rb", "Sr", "Y",  "Zr", "Nb", "Mo", "Tc", "Ru", "Rh", "Pd", "Ag", "Cd",
  "In", "Sn", "Sb", "Te", "I",  "Xe", "Cs", "Ba", "La", "Ce", "Pr", "Nd", "Pm", "Sm", "Eu", "Gd",
  "Tb", "Dy", "Ho", "Er", "Tm", "Yb", "Lu", "Hf", "Ta", "W",  "Re", "Os", "Ir", "Pt", "Au", "Hg",
  "Tl", "Pb", "Bi", "Po", "At", "Rn", "Fr", "Ra", "Ac", "Th", "Pa", "U",  "Np", "Pu"};

static inline bool check_sublayer(const char* element)
{
  return strcmp(element, "Mo") == 0 || strcmp(element, "S") == 0 ||
         strcmp(element, "Se") == 0 || strcmp(element, "W") == 0 ||
         strcmp(element, "Te");
}

ILP_NEP::ILP_NEP(FILE* fid_ilp, FILE* fid_nep_map, int num_types, int num_atoms)
{
  // read ILP elements
  printf("Use %d-element ILP potential with elements:\n", num_types);
  if (!(num_types >= 1 && num_types <= MAX_TYPE_ILP_NEP)) {
    PRINT_INPUT_ERROR("Incorrect type number of ILP parameters.\n");
  }
  std::vector<std::string> ilp_elements(num_types);
  for (int n = 0; n < num_types; ++n) {
    char atom_symbol[10];
    int count = fscanf(fid_ilp, "%s", atom_symbol);
    PRINT_SCANF_ERROR(count, 1, "Reading error for ILP potential.");
    printf(" %s", atom_symbol);
    ilp_elements[n] = atom_symbol;
    sublayer_flag[n] = check_sublayer(atom_symbol);
  }
  printf("\n");

  // read ILP group method
  PRINT_SCANF_ERROR(fscanf(fid_ilp, "%d", &ilp_group_method), 1, 
  "Reading error for ILP group method.");
  printf("Use group method %d to identify molecule for ILP.\n", ilp_group_method);

  // read ILP parameters
  float beta, alpha, delta, epsilon, C, d, sR;
  float reff, C6, S, rcut_ilp, rcut_global;
  rc = 0.0;
  for (int n = 0; n < num_types; ++n) {
    for (int m = 0; m < num_types; ++m) {
      int count = fscanf(fid_ilp, "%f%f%f%f%f%f%f%f%f%f%f%f", \
      &beta, &alpha, &delta, &epsilon, &C, &d, &sR, &reff, &C6, &S, \
      &rcut_ilp, &rcut_global);
      PRINT_SCANF_ERROR(count, 12, "Reading error for ILP potential.");

      ilp_para.C[n][m] = C;
      ilp_para.C_6[n][m] = C6;
      ilp_para.d[n][m] = d;
      ilp_para.d_Seff[n][m] = d / sR / reff;
      ilp_para.epsilon[n][m] = epsilon;
      ilp_para.z0[n][m] = beta;
      ilp_para.lambda[n][m] = alpha / beta;
      ilp_para.delta2inv[n][m] = 1.0 / (delta * delta);
      ilp_para.S[n][m] = S;
      ilp_para.rcutsq_ilp[n][m] = rcut_ilp * rcut_ilp;
      ilp_para.rcut_global[n][m] = rcut_global;
      float meV = 1e-3 * S;
      ilp_para.C[n][m] *= meV;
      ilp_para.C_6[n][m] *= meV;
      ilp_para.epsilon[n][m] *= meV;

      if (rc < rcut_global)
        rc = rcut_global;
    }
  }

  // read NEP group method from nep map file
  PRINT_SCANF_ERROR(fscanf(fid_nep_map, "%d", &nep_group_method), 1, 
  "Reading error for NEP group method.");
  printf("Use group method %d to identify molecule for NEP.\n", nep_group_method);

  // read the number of NEP file
  PRINT_SCANF_ERROR(fscanf(fid_nep_map, "%d", &num_nep), 1, 
  "Reading error for the number of NEP file.");
  printf("NEP file number: %d\n", num_nep);

  // init parameter vectors
  parambs.resize(num_nep);
  annmbs.resize(num_nep);
  nep_data.parameters.resize(num_nep);

  // init type map cpu
  type_map_cpu.resize(num_types * num_nep, -1);
  
  // read NEP parameter from each NEP file
  for (int i = 0; i < num_nep; ++i) {
    printf("\nReading NEP %d.\n", i);
    char nep_file[100];
    int count = fscanf(fid_nep_map, "%s", nep_file);
    PRINT_SCANF_ERROR(count, 1, "reading error for NEP filename");

    std::ifstream input(nep_file);
    if (!input.is_open()) {
      std::cout << "Failed to open " << nep_file << std::endl;
      exit(1);
    }

    // nep3 1 C
    std::vector<std::string> tokens = get_tokens(input);
    if (tokens.size() < 3) {
      std::cout << "The first line of nep.txt should have at least 3 items." << std::endl;
      exit(1);
    }
    if (tokens[0] == "nep3") {
      parambs[i].version = 3;
    } else if (tokens[0] == "nep4") {
      parambs[i].version = 4;
    } else if (tokens[0] == "nep5") {
      parambs[i].version = 5;
    } else {
      std::cout << tokens[0]
                << " is an unsupported NEP model. We only support NEP3 and NEP4 models now."
                << std::endl;
      exit(1);
    }
    parambs[i].num_types = get_int_from_token(tokens[1], __FILE__, __LINE__);
    if (tokens.size() != 2 + parambs[i].num_types) {
      std::cout << "The first line of nep.txt should have " << parambs[i].num_types << " atom symbols."
                << std::endl;
      exit(1);
    }

    if (parambs[i].num_types == 1) {
      printf("Use the NEP%d potential with %d atom type.\n", parambs[i].version, parambs[i].num_types);
    } else {
      printf("Use the NEP%d potential with %d atom types.\n", parambs[i].version, parambs[i].num_types);
    }

    for (int n = 0; n < parambs[i].num_types; ++n) {
      int atomic_number = 0;
      for (int m = 0; m < NUM_ELEMENTS; ++m) {
        if (tokens[2 + n] == ELEMENTS[m]) {
          atomic_number = m + 1;
          break;
        }
      }
      parambs[i].atomic_numbers[n] = atomic_number - 1;
      printf("    type %d (%s with Z = %d).\n", n, tokens[2 + n].c_str(), atomic_number);

      // update type map
      // for example: if ilp elements is C N B, element in nep 0 is C, elements in nep 1 are B N
      // type map should be [0,    -1,    -1,    -1,    1,    0]
      //   ilp element       C      N      B      C     N     B
      //   nep 0 element     C(0)   null   null
      //   nep 1 element                          null  N(1)  B(0)
      for (int m = 0; m < num_types; ++m) {
        if (tokens[2 + n] == ilp_elements[m]) {
          type_map_cpu[m + i * num_types] = n;
        }
      }
    }

    // cutoff 4.2 3.7 80 47 1
    tokens = get_tokens(input);
    if (tokens.size() != 5 && tokens.size() != 8) {
      std::cout << "This line should be cutoff rc_radial rc_angular MN_radial MN_angular "
                   "[radial_factor] [angular_factor] [zbl_factor].\n";
      exit(1);
    }
    parambs[i].rc_radial = get_float_from_token(tokens[1], __FILE__, __LINE__);
    parambs[i].rc_angular = get_float_from_token(tokens[2], __FILE__, __LINE__);
    printf("    radial cutoff = %g A.\n", parambs[i].rc_radial);
    printf("    angular cutoff = %g A.\n", parambs[i].rc_angular);

    int MN_radial = get_int_from_token(tokens[3], __FILE__, __LINE__);
    int MN_angular = get_int_from_token(tokens[4], __FILE__, __LINE__);
    printf("    MN_radial = %d.\n", MN_radial);
    if (MN_radial > 819) {
      std::cout << "The maximum number of neighbors exceeds 819. Please reduce this value."
                << std::endl;
      exit(1);
    }
    parambs[i].MN_radial = int(ceil(MN_radial * 1.25));
    parambs[i].MN_angular = int(ceil(MN_angular * 1.25));
    max_MN_radial = max(max_MN_radial, parambs[i].MN_radial);
    max_MN_angular = max(max_MN_angular, parambs[i].MN_angular);
    printf("    enlarged MN_radial = %d.\n", parambs[i].MN_radial);
    printf("    enlarged MN_angular = %d.\n", parambs[i].MN_angular);

    if (tokens.size() == 8) {
      parambs[i].typewise_cutoff_radial_factor = get_float_from_token(tokens[5], __FILE__, __LINE__);
      parambs[i].typewise_cutoff_angular_factor = get_float_from_token(tokens[6], __FILE__, __LINE__);
      if (parambs[i].typewise_cutoff_radial_factor > 0.0f) {
        parambs[i].use_typewise_cutoff = true;
      }
    }
  #ifdef USE_TABLE
    if (paramb.use_typewise_cutoff) {
      PRINT_INPUT_ERROR("Cannot use tabulated radial functions with typewise cutoff.");
    }
  #endif

    // n_max 10 8
    tokens = get_tokens(input);
    if (tokens.size() != 3) {
      std::cout << "This line should be n_max n_max_radial n_max_angular." << std::endl;
      exit(1);
    }
    parambs[i].n_max_radial = get_int_from_token(tokens[1], __FILE__, __LINE__);
    parambs[i].n_max_angular = get_int_from_token(tokens[2], __FILE__, __LINE__);
    max_n_max_angular = max(max_n_max_angular, parambs[i].n_max_angular);
    printf("    n_max_radial = %d.\n", parambs[i].n_max_radial);
    printf("    n_max_angular = %d.\n", parambs[i].n_max_angular);

    // basis_size 10 8
    tokens = get_tokens(input);
    if (tokens.size() != 3) {
      std::cout << "This line should be basis_size basis_size_radial basis_size_angular."
                << std::endl;
      exit(1);
    }
    parambs[i].basis_size_radial = get_int_from_token(tokens[1], __FILE__, __LINE__);
    parambs[i].basis_size_angular = get_int_from_token(tokens[2], __FILE__, __LINE__);
    printf("    basis_size_radial = %d.\n", parambs[i].basis_size_radial);
    printf("    basis_size_angular = %d.\n", parambs[i].basis_size_angular);

    // l_max
    tokens = get_tokens(input);
    if (tokens.size() != 4) {
      std::cout << "This line should be l_max l_max_3body l_max_4body l_max_5body." << std::endl;
      exit(1);
    }

    parambs[i].L_max = get_int_from_token(tokens[1], __FILE__, __LINE__);
    printf("    l_max_3body = %d.\n", parambs[i].L_max);
    parambs[i].num_L = parambs[i].L_max;

    int L_max_4body = get_int_from_token(tokens[2], __FILE__, __LINE__);
    int L_max_5body = get_int_from_token(tokens[3], __FILE__, __LINE__);
    printf("    l_max_4body = %d.\n", L_max_4body);
    printf("    l_max_5body = %d.\n", L_max_5body);
    if (L_max_4body == 2) {
      parambs[i].num_L += 1;
    }
    if (L_max_5body == 1) {
      parambs[i].num_L += 1;
    }

    parambs[i].dim_angular = (parambs[i].n_max_angular + 1) * parambs[i].num_L;

    // ANN
    tokens = get_tokens(input);
    if (tokens.size() != 3) {
      std::cout << "This line should be ANN num_neurons 0." << std::endl;
      exit(1);
    }
    annmbs[i].num_neurons1 = get_int_from_token(tokens[1], __FILE__, __LINE__);
    annmbs[i].dim = (parambs[i].n_max_radial + 1) + parambs[i].dim_angular;
    nep_model_type = parambs[i].model_type;
    if (parambs[i].model_type == 3) {
      annmbs[i].dim += 1;
    }
    max_dim = max(max_dim, annmbs[i].dim);
    printf("    ANN = %d-%d-1.\n", annmbs[i].dim, annmbs[i].num_neurons1);

    // calculated parameters:
    parambs[i].rcinv_radial = 1.0f / parambs[i].rc_radial;
    parambs[i].rcinv_angular = 1.0f / parambs[i].rc_angular;
    parambs[i].num_types_sq = parambs[i].num_types * parambs[i].num_types;

    if (parambs[i].version == 3) {
      annmbs[i].num_para_ann = (annmbs[i].dim + 2) * annmbs[i].num_neurons1 + 1;
    } else if (parambs[i].version == 4) {
      annmbs[i].num_para_ann = (annmbs[i].dim + 2) * annmbs[i].num_neurons1 * parambs[i].num_types + 1;
    } else {
      annmbs[i].num_para_ann = ((annmbs[i].dim + 2) * annmbs[i].num_neurons1 + 1) * parambs[i].num_types + 1;
    }
    if (parambs[i].model_type == 2) {
      // Polarizability models have twice as many parameters
      annmbs[i].num_para_ann *= 2;
    }
    printf("    number of neural network parameters = %d.\n", annmbs[i].num_para_ann);
    int num_para_descriptor =
      parambs[i].num_types_sq * ((parambs[i].n_max_radial + 1) * (parambs[i].basis_size_radial + 1) +
                             (parambs[i].n_max_angular + 1) * (parambs[i].basis_size_angular + 1));
    printf("    number of descriptor parameters = %d.\n", num_para_descriptor);
    annmbs[i].num_para = annmbs[i].num_para_ann + num_para_descriptor;
    printf("    total number of parameters = %d.\n", annmbs[i].num_para);

    parambs[i].num_c_radial =
      parambs[i].num_types_sq * (parambs[i].n_max_radial + 1) * (parambs[i].basis_size_radial + 1);

    // NN and descriptor parameters
    std::vector<float> parameters(annmbs[i].num_para);
    for (int n = 0; n < annmbs[i].num_para; ++n) {
      tokens = get_tokens(input);
      parameters[n] = get_float_from_token(tokens[0], __FILE__, __LINE__);
    }
    nep_data.parameters[i].resize(annmbs[i].num_para);
    nep_data.parameters[i].copy_from_host(parameters.data());
    update_potential(nep_data.parameters[i].data(), parambs[i], annmbs[i]);
    for (int d = 0; d < annmbs[i].dim; ++d) {
      tokens = get_tokens(input);
      parambs[i].q_scaler[d] = get_float_from_token(tokens[0], __FILE__, __LINE__);
    }

  }

  // read nep map to identify the nep for each group
  int num_nep_group = 0;
  PRINT_SCANF_ERROR(fscanf(fid_nep_map, "%d", &num_nep_group), 1, 
  "Reading error for the number of nep group.");
  nep_map_cpu.resize(num_nep_group);
  for (int i = 0; i < num_nep_group; ++i) {
    int nep_i = 0;    // which nep this group use
    int count = fscanf(fid_nep_map, "%d", &nep_i);
    PRINT_SCANF_ERROR(count, 1, "reading error for nep number of group.");
    if (nep_i >= num_nep) {
      if (num_nep == 1) {
        printf("There is only 1 nep file, but you set group %d of group method %d \
        to nep %d", i, nep_group_method, nep_i);
      } else {
        printf("There are %d nep files, but you set group %d of group method %d \
        to nep %d", num_nep, i, nep_group_method, nep_i);
      }
      exit(1);
    }
    nep_map_cpu[i] = nep_i;
    printf("group %d uses NEP %d.\n", i, nep_i);
  }

  // cp two maps to gpu
  nep_map.resize(num_nep_group);
  type_map.resize(num_types * num_nep);
  nep_map.copy_from_host(nep_map_cpu.data());
  type_map.copy_from_host(type_map_cpu.data());


  // initialize ilp neighbor lists and some temp vectors
  int max_neighbor_number = min(num_atoms, CUDA_MAX_NL_ILP_NEP_CBN);
  ilp_data.NN.resize(num_atoms);
  ilp_data.NL.resize(num_atoms * max_neighbor_number);
  ilp_data.cell_count.resize(num_atoms);
  ilp_data.cell_count_sum.resize(num_atoms);
  ilp_data.cell_contents.resize(num_atoms);

  // init ilp neighbor list
  ilp_data.ilp_NN.resize(num_atoms);
  ilp_data.ilp_NL.resize(num_atoms * MAX_ILP_NEIGHBOR_CBN);
  ilp_data.reduce_NL.resize(num_atoms * max_neighbor_number);
  ilp_data.big_ilp_NN.resize(num_atoms);
  ilp_data.big_ilp_NL.resize(num_atoms * MAX_BIG_ILP_NEIGHBOR_CBN);

  ilp_data.f12x.resize(num_atoms * max_neighbor_number);
  ilp_data.f12y.resize(num_atoms * max_neighbor_number);
  ilp_data.f12z.resize(num_atoms * max_neighbor_number);

  ilp_data.f12x_ilp_neigh.resize(num_atoms * MAX_ILP_NEIGHBOR_CBN);
  ilp_data.f12y_ilp_neigh.resize(num_atoms * MAX_ILP_NEIGHBOR_CBN);
  ilp_data.f12z_ilp_neigh.resize(num_atoms * MAX_ILP_NEIGHBOR_CBN);

  // init constant cutoff coeff
  float h_tap_coeff[8] = \
    {1.0f, 0.0f, 0.0f, 0.0f, -35.0f, 84.0f, -70.0f, 20.0f};
  CHECK(gpuMemcpyToSymbol(Tap_coeff, h_tap_coeff, 8 * sizeof(float)));

  // set ilp_flag to 1
  ilp_flag = 1;

  // initialize nep neighbor lists
  nep_data.f12x.resize(num_atoms * max_MN_angular);
  nep_data.f12y.resize(num_atoms * max_MN_angular);
  nep_data.f12z.resize(num_atoms * max_MN_angular);
  nep_data.NN_radial.resize(num_atoms);
  nep_data.NL_radial.resize(num_atoms * max_MN_radial);
  nep_data.NN_angular.resize(num_atoms);
  nep_data.NL_angular.resize(num_atoms * max_MN_angular);
  nep_data.Fp.resize(num_atoms * max_dim);
  nep_data.sum_fxyz.resize(num_atoms * (max_n_max_angular + 1) * NUM_OF_ABC);
  nep_data.cell_count.resize(num_atoms);
  nep_data.cell_count_sum.resize(num_atoms);
  nep_data.cell_contents.resize(num_atoms);
  nep_data.cpu_NN_radial.resize(num_atoms);
  nep_data.cpu_NN_angular.resize(num_atoms);

#ifdef USE_TABLE
  construct_table(parameters.data());
  printf("    use tabulated radial functions to speed up.\n");
#endif

}

void ILP_NEP::update_potential(float* parameters, ParaMB& paramb, ANN& ann)
{
  float* pointer = parameters;
  for (int t = 0; t < paramb.num_types; ++t) {
    if (t > 0 && paramb.version == 3) { // Use the same set of NN parameters for NEP3
      pointer -= (ann.dim + 2) * ann.num_neurons1;
    }
    ann.w0[t] = pointer;
    pointer += ann.num_neurons1 * ann.dim;
    ann.b0[t] = pointer;
    pointer += ann.num_neurons1;
    ann.w1[t] = pointer;
    pointer += ann.num_neurons1;
    if (paramb.version == 5) {
      pointer += 1; // one extra bias for NEP5 stored in ann.w1[t]
    }
  }
  ann.b1 = pointer;
  pointer += 1;

  ann.c = pointer;
}

ILP_NEP::~ILP_NEP(void)
{
  // nothing
}

static __device__ __forceinline__ float calc_Tap(const float r_ij, const float Rcutinv)
{
  float Tap, r;

  r = r_ij * Rcutinv;
  Tap = Tap_coeff[7];
  Tap = Tap * r + Tap_coeff[6];
  Tap = Tap * r + Tap_coeff[5];
  Tap = Tap * r + Tap_coeff[4];
  Tap = Tap * r + Tap_coeff[3];
  Tap = Tap * r + Tap_coeff[2];
  Tap = Tap * r + Tap_coeff[1];
  Tap = Tap * r + Tap_coeff[0];

  return Tap;
}

// calculate the derivatives of long-range cutoff term
static __device__ __forceinline__ float calc_dTap(const float r_ij, const float Rcutinv)
{
  float dTap, r;
  
  r = r_ij * Rcutinv;
  dTap = 7.0f * Tap_coeff[7];
  dTap = dTap * r + 6.0f * Tap_coeff[6];
  dTap = dTap * r + 5.0f * Tap_coeff[5];
  dTap = dTap * r + 4.0f * Tap_coeff[4];
  dTap = dTap * r + 3.0f * Tap_coeff[3];
  dTap = dTap * r + 2.0f * Tap_coeff[2];
  dTap = dTap * r + Tap_coeff[1];
  dTap *= Rcutinv;

  return dTap;
}

// create ILP neighbor list from main neighbor list to calculate normals
static __global__ void ILP_neighbor(
  const int number_of_particles,
  const int N1,
  const int N2,
  const Box box,
  const int *g_neighbor_number,
  const int *g_neighbor_list,
  const int *g_type,
  ILP_Para ilp_para,
  const double* __restrict__ g_x,
  const double* __restrict__ g_y,
  const double* __restrict__ g_z,
  int *ilp_neighbor_number,
  int *ilp_neighbor_list,
  const int *group_label,
  bool sublayer_flag[MAX_TYPE_ILP_NEP])
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1; // particle index

  if (n1 < N2) {
    int neighptr[10], check[10], neighsort[10];
    for (int ll = 0; ll < 10; ++ll) {
      neighptr[ll] = -1;
      neighsort[ll] = -1;
      check[ll] = -1;
    }

    int count = 0;
    int neighbor_number = g_neighbor_number[n1];
    int type1 = g_type[n1];
    double x1 = g_x[n1];
    double y1 = g_y[n1];
    double z1 = g_z[n1];

    for (int i1 = 0; i1 < neighbor_number; ++i1) {
      int n2 = g_neighbor_list[n1 + number_of_particles * i1];
      int type2 = g_type[n2];

      double x12 = g_x[n2] - x1;
      double y12 = g_y[n2] - y1;
      double z12 = g_z[n2] - z1;
      apply_mic(box, x12, y12, z12);
      double d12sq = x12 * x12 + y12 * y12 + z12 * z12;
      double rcutsq = ilp_para.rcutsq_ilp[type1][type2];

      // if material has sublayer, calc normal in sublayer (same type)
      if (group_label[n1] == group_label[n2] && d12sq < rcutsq && 
          (type1 == type2 || !sublayer_flag[type1]) && d12sq != 0) {
        // ilp_neighbor_list[count++ * number_of_particles + n1] = n2;
        neighptr[count++] = n2;
      }
    }

    // sort orders of neighbors
    if (sublayer_flag[type1]) {
      // init neighsort
      for (int ll = 0; ll < count; ++ll) {
        neighsort[ll] = neighptr[ll];
        check[ll] = neighptr[ll];
      }

      // select the first neighbor of atom n1
      if (count == MAX_ILP_NEIGHBOR_TMD) {
        neighsort[0] = neighptr[0];
        check[0] = -1;
      } else if (count < MAX_ILP_NEIGHBOR_TMD && count > 0) {
        for (int jj = 0; jj < count; ++jj) {
          int j = neighptr[jj];
          int jtype = g_type[j];
          int count_temp = 0;
          for (int ll = 0; ll < count; ++ll) {
            int l = neighptr[ll];
            int ltype = g_type[l];
            if (l == j) continue;
            double deljx = g_x[l] - g_x[j];
            double deljy = g_y[l] - g_y[j];
            double deljz = g_z[l] - g_z[j];
            apply_mic(box, deljx, deljy, deljz);
            double rsqlj = deljx * deljx + deljy * deljy + deljz * deljz;
            if (rsqlj != 0 && rsqlj < ilp_para.rcutsq_ilp[ltype][jtype]) {
              ++count_temp;
            }
          }
          if (count_temp == 1) {
            neighsort[0] = neighptr[jj];
            check[jj] = -1;
            break;
          }
        }
      } else if (count > MAX_ILP_NEIGHBOR_TMD) {
        printf("ERROR in ILP NEIGHBOR LIST\n");
        printf("\n===== ILP neighbor number[%d] is greater than 6 =====\n", count);
        exit(1);
      }

      // sort the order of neighbors of atom n1
      for (int jj = 0; jj < count; ++jj) {
        int j = neighsort[jj];
        int jtype = g_type[j];
        int ll = 0;
        while (ll < count) {
          int l = neighptr[ll];
          if (check[ll] == -1) {
            ++ll;
            continue;
          }
          int ltype = g_type[l];
          double deljx = g_x[l] - g_x[j];
          double deljy = g_y[l] - g_y[j];
          double deljz = g_z[l] - g_z[j];
          apply_mic(box, deljx, deljy, deljz);
          double rsqlj = deljx * deljx + deljy * deljy + deljz * deljz;

          if (abs(rsqlj) >= 1e-6 && rsqlj < ilp_para.rcutsq_ilp[ltype][jtype]) {
            neighsort[jj + 1] = l;
            check[ll] = -1;
            break;
          }
          ++ll;
        }
      }
    }

    ilp_neighbor_number[n1] = count;
    for (int jj = 0; jj < count; ++jj) {
      ilp_neighbor_list[jj * number_of_particles + n1] = neighsort[jj];
    }
  }
}

// modulo func to change atom index
static __device__ __forceinline__ int modulo(int k, int range)
{
  return (k + range) % range;
}

// calculate the normals and its derivatives for C B N
static __device__ void calc_normal_cbn(
  float (&vet)[MAX_ILP_NEIGHBOR_TMD][3],
  int cont,
  float (&normal)[3],
  float (&dnormdri)[3][3],
  float (&dnormal)[3][MAX_ILP_NEIGHBOR_TMD][3])
{
  int id, ip, m;
  float pv12[3], pv31[3], pv23[3], n1[3], dni[3];
  float dnn[3][3], dpvdri[3][3];
  float dn1[3][3][3], dpv12[3][3][3], dpv23[3][3][3], dpv31[3][3][3];

  float nninv, continv;

  // initialize the arrays
  for (id = 0; id < 3; id++) {
    pv12[id] = 0.0f;
    pv31[id] = 0.0f;
    pv23[id] = 0.0f;
    n1[id] = 0.0f;
    dni[id] = 0.0f;
    for (ip = 0; ip < 3; ip++) {
      dnn[ip][id] = 0.0f;
      dpvdri[ip][id] = 0.0f;
      for (m = 0; m < 3; m++) {
        dpv12[ip][id][m] = 0.0f;
        dpv31[ip][id][m] = 0.0f;
        dpv23[ip][id][m] = 0.0f;
        dn1[ip][id][m] = 0.0f;
      }
    }
  }

  if (cont <= 1) {
    normal[0] = 0.0;
    normal[1] = 0.0;
    normal[2] = 1.0;
    for (id = 0; id < 3; ++id) {
      for (ip = 0; ip < 3; ++ip) {
        dnormdri[id][ip] = 0.0;
        for (m = 0; m < 3; ++m) {
          dnormal[id][ip][m] = 0.0;
        }
      }
    }
  } else if (cont == 2) {
    pv12[0] = vet[0][1] * vet[1][2] - vet[1][1] * vet[0][2];
    pv12[1] = vet[0][2] * vet[1][0] - vet[1][2] * vet[0][0];
    pv12[2] = vet[0][0] * vet[1][1] - vet[1][0] * vet[0][1];
    // derivatives of pv12[0] to ri
    dpvdri[0][0] = 0.0f;
    dpvdri[0][1] = vet[0][2] - vet[1][2];
    dpvdri[0][2] = vet[1][1] - vet[0][1];
    // derivatives of pv12[1] to ri
    dpvdri[1][0] = vet[1][2] - vet[0][2];
    dpvdri[1][1] = 0.0f;
    dpvdri[1][2] = vet[0][0] - vet[1][0];
    // derivatives of pv12[2] to ri
    dpvdri[2][0] = vet[0][1] - vet[1][1];
    dpvdri[2][1] = vet[1][0] - vet[0][0];
    dpvdri[2][2] = 0.0f;

    dpv12[0][0][0] = 0.0f;
    dpv12[0][1][0] = vet[1][2];
    dpv12[0][2][0] = -vet[1][1];
    dpv12[1][0][0] = -vet[1][2];
    dpv12[1][1][0] = 0.0f;
    dpv12[1][2][0] = vet[1][0];
    dpv12[2][0][0] = vet[1][1];
    dpv12[2][1][0] = -vet[1][0];
    dpv12[2][2][0] = 0.0f;

    // derivatives respect to the second neighbor, atom l
    dpv12[0][0][1] = 0.0f;
    dpv12[0][1][1] = -vet[0][2];
    dpv12[0][2][1] = vet[0][1];
    dpv12[1][0][1] = vet[0][2];
    dpv12[1][1][1] = 0.0f;
    dpv12[1][2][1] = -vet[0][0];
    dpv12[2][0][1] = -vet[0][1];
    dpv12[2][1][1] = vet[0][0];
    dpv12[2][2][1] = 0.0f;

    // derivatives respect to the third neighbor, atom n
    // derivatives of pv12 to rn is zero
    for (id = 0; id < 3; id++) {
      for (ip = 0; ip < 3; ip++) { dpv12[id][ip][2] = 0.0f; }
    }

    n1[0] = pv12[0];
    n1[1] = pv12[1];
    n1[2] = pv12[2];
    // the magnitude of the normal vector
    // nn2 = n1[0] * n1[0] + n1[1] * n1[1] + n1[2] * n1[2];
    // nn = sqrt(nn2);
    // nninv = 1.0 / nn;
    nninv = rnorm3df(n1[0], n1[1], n1[2]);
    
    // TODO
    // if (nn == 0) error->one(FLERR, "The magnitude of the normal vector is zero");
    // the unit normal vector
    normal[0] = n1[0] * nninv;
    normal[1] = n1[1] * nninv;
    normal[2] = n1[2] * nninv;
    // derivatives of nn, dnn:3x1 vector
    dni[0] = (n1[0] * dpvdri[0][0] + n1[1] * dpvdri[1][0] + n1[2] * dpvdri[2][0]) * nninv;
    dni[1] = (n1[0] * dpvdri[0][1] + n1[1] * dpvdri[1][1] + n1[2] * dpvdri[2][1]) * nninv;
    dni[2] = (n1[0] * dpvdri[0][2] + n1[1] * dpvdri[1][2] + n1[2] * dpvdri[2][2]) * nninv;
    // derivatives of unit vector ni respect to ri, the result is 3x3 matrix
    for (id = 0; id < 3; id++) {
      for (ip = 0; ip < 3; ip++) {
        dnormdri[id][ip] = dpvdri[id][ip] * nninv - n1[id] * dni[ip] * nninv * nninv;
      }
    }
    // derivatives of non-normalized normal vector, dn1:3x3x3 array
    for (id = 0; id < 3; id++) {
      for (ip = 0; ip < 3; ip++) {
        for (m = 0; m < 3; m++) { dn1[id][ip][m] = dpv12[id][ip][m]; }
      }
    }
    // derivatives of nn, dnn:3x3 vector
    // dnn[id][m]: the derivative of nn respect to r[id][m], id,m=0,1,2
    // r[id][m]: the id's component of atom m
    for (m = 0; m < 3; m++) {
      for (id = 0; id < 3; id++) {
        dnn[id][m] = (n1[0] * dn1[0][id][m] + n1[1] * dn1[1][id][m] + n1[2] * dn1[2][id][m]) * nninv;
      }
    }
    // dnormal[id][ip][m][i]: the derivative of normal[id] respect to r[ip][m], id,ip=0,1,2
    // for atom m, which is a neighbor atom of atom i, m=0,jnum-1
    for (m = 0; m < 3; m++) {
      for (id = 0; id < 3; id++) {
        for (ip = 0; ip < 3; ip++) {
          dnormal[id][ip][m] = dn1[id][ip][m] * nninv - n1[id] * dnn[ip][m] * nninv * nninv;
        }
      }
    }
  } else if (cont == 3) {
    continv = 1.0 / cont;

    pv12[0] = vet[0][1] * vet[1][2] - vet[1][1] * vet[0][2];
    pv12[1] = vet[0][2] * vet[1][0] - vet[1][2] * vet[0][0];
    pv12[2] = vet[0][0] * vet[1][1] - vet[1][0] * vet[0][1];
    // derivatives respect to the first neighbor, atom k
    dpv12[0][0][0] = 0.0f;
    dpv12[0][1][0] = vet[1][2];
    dpv12[0][2][0] = -vet[1][1];
    dpv12[1][0][0] = -vet[1][2];
    dpv12[1][1][0] = 0.0f;
    dpv12[1][2][0] = vet[1][0];
    dpv12[2][0][0] = vet[1][1];
    dpv12[2][1][0] = -vet[1][0];
    dpv12[2][2][0] = 0.0f;
    // derivatives respect to the second neighbor, atom l
    dpv12[0][0][1] = 0.0f;
    dpv12[0][1][1] = -vet[0][2];
    dpv12[0][2][1] = vet[0][1];
    dpv12[1][0][1] = vet[0][2];
    dpv12[1][1][1] = 0.0f;
    dpv12[1][2][1] = -vet[0][0];
    dpv12[2][0][1] = -vet[0][1];
    dpv12[2][1][1] = vet[0][0];
    dpv12[2][2][1] = 0.0f;

    // derivatives respect to the third neighbor, atom n
    for (id = 0; id < 3; id++) {
      for (ip = 0; ip < 3; ip++) { dpv12[id][ip][2] = 0.0f; }
    }

    pv31[0] = vet[2][1] * vet[0][2] - vet[0][1] * vet[2][2];
    pv31[1] = vet[2][2] * vet[0][0] - vet[0][2] * vet[2][0];
    pv31[2] = vet[2][0] * vet[0][1] - vet[0][0] * vet[2][1];
    // derivatives respect to the first neighbor, atom k
    dpv31[0][0][0] = 0.0f;
    dpv31[0][1][0] = -vet[2][2];
    dpv31[0][2][0] = vet[2][1];
    dpv31[1][0][0] = vet[2][2];
    dpv31[1][1][0] = 0.0f;
    dpv31[1][2][0] = -vet[2][0];
    dpv31[2][0][0] = -vet[2][1];
    dpv31[2][1][0] = vet[2][0];
    dpv31[2][2][0] = 0.0f;
    // derivatives respect to the third neighbor, atom n
    dpv31[0][0][2] = 0.0f;
    dpv31[0][1][2] = vet[0][2];
    dpv31[0][2][2] = -vet[0][1];
    dpv31[1][0][2] = -vet[0][2];
    dpv31[1][1][2] = 0.0f;
    dpv31[1][2][2] = vet[0][0];
    dpv31[2][0][2] = vet[0][1];
    dpv31[2][1][2] = -vet[0][0];
    dpv31[2][2][2] = 0.0f;
    // derivatives respect to the second neighbor, atom l
    for (id = 0; id < 3; id++) {
      for (ip = 0; ip < 3; ip++) { dpv31[id][ip][1] = 0.0f; }
    }

    pv23[0] = vet[1][1] * vet[2][2] - vet[2][1] * vet[1][2];
    pv23[1] = vet[1][2] * vet[2][0] - vet[2][2] * vet[1][0];
    pv23[2] = vet[1][0] * vet[2][1] - vet[2][0] * vet[1][1];
    // derivatives respect to the second neighbor, atom k
    for (id = 0; id < 3; id++) {
      for (ip = 0; ip < 3; ip++) { dpv23[id][ip][0] = 0.0f; }
    }
    // derivatives respect to the second neighbor, atom l
    dpv23[0][0][1] = 0.0f;
    dpv23[0][1][1] = vet[2][2];
    dpv23[0][2][1] = -vet[2][1];
    dpv23[1][0][1] = -vet[2][2];
    dpv23[1][1][1] = 0.0f;
    dpv23[1][2][1] = vet[2][0];
    dpv23[2][0][1] = vet[2][1];
    dpv23[2][1][1] = -vet[2][0];
    dpv23[2][2][1] = 0.0f;
    // derivatives respect to the third neighbor, atom n
    dpv23[0][0][2] = 0.0f;
    dpv23[0][1][2] = -vet[1][2];
    dpv23[0][2][2] = vet[1][1];
    dpv23[1][0][2] = vet[1][2];
    dpv23[1][1][2] = 0.0f;
    dpv23[1][2][2] = -vet[1][0];
    dpv23[2][0][2] = -vet[1][1];
    dpv23[2][1][2] = vet[1][0];
    dpv23[2][2][2] = 0.0f;

    //############################################################################################
    // average the normal vectors by using the 3 neighboring planes
    n1[0] = (pv12[0] + pv31[0] + pv23[0]) * continv;
    n1[1] = (pv12[1] + pv31[1] + pv23[1]) * continv;
    n1[2] = (pv12[2] + pv31[2] + pv23[2]) * continv;
    // the magnitude of the normal vector
    // nn2 = n1[0] * n1[0] + n1[1] * n1[1] + n1[2] * n1[2];
    // nn = sqrt(nn2);

    // nninv = 1.0 / nn;
    nninv = rnorm3df(n1[0], n1[1], n1[2]);
    // TODO
    // if (nn == 0) error->one(FLERR, "The magnitude of the normal vector is zero");
    // the unit normal vector
    normal[0] = n1[0] * nninv;
    normal[1] = n1[1] * nninv;
    normal[2] = n1[2] * nninv;

    // for the central atoms, dnormdri is always zero
    for (id = 0; id < 3; id++) {
      for (ip = 0; ip < 3; ip++) { dnormdri[id][ip] = 0.0f; }
    }

    // derivatives of non-normalized normal vector, dn1:3x3x3 array
    for (id = 0; id < 3; id++) {
      for (ip = 0; ip < 3; ip++) {
        for (m = 0; m < 3; m++) {
          dn1[id][ip][m] = (dpv12[id][ip][m] + dpv23[id][ip][m] + dpv31[id][ip][m]) * continv;
        }
      }
    }
    // derivatives of nn, dnn:3x3 vector
    // dnn[id][m]: the derivative of nn respect to r[id][m], id,m=0,1,2
    // r[id][m]: the id's component of atom m
    for (m = 0; m < 3; m++) {
      for (id = 0; id < 3; id++) {
        dnn[id][m] = (n1[0] * dn1[0][id][m] + n1[1] * dn1[1][id][m] + n1[2] * dn1[2][id][m]) * nninv;
      }
    }
    // dnormal[id][ip][m][i]: the derivative of normal[id] respect to r[ip][m], id,ip=0,1,2
    // for atom m, which is a neighbor atom of atom i, m=0,jnum-1
    for (m = 0; m < 3; m++) {
      for (id = 0; id < 3; id++) {
        for (ip = 0; ip < 3; ip++) {
          dnormal[id][ip][m] = dn1[id][ip][m] * nninv - n1[id] * dnn[ip][m] * nninv * nninv;
        }
      }
    }
  } else {
    // TODO: error! too many neighbors for calculating normals
  }
}

// calculate the normals and its derivatives for TMDs
static __device__ void calc_normal_tmd(
  float (&vect)[MAX_ILP_NEIGHBOR_TMD][3],
  int cont,
  float (&normal)[3],
  float (&dnormdri)[3][3],
  float (&dnormal)[3][MAX_ILP_NEIGHBOR_TMD][3])
{
  int id, ip, m;
  float  dni[3];
  float  dnn[3][3], dpvdri[3][3];
  float Nave[3], pvet[MAX_ILP_NEIGHBOR_TMD][3], dpvet1[MAX_ILP_NEIGHBOR_TMD][3][3], dpvet2[MAX_ILP_NEIGHBOR_TMD][3][3], dNave[3][MAX_ILP_NEIGHBOR_TMD][3];

  float nninv;

  // initialize the arrays
  for (id = 0; id < 3; id++) {
    dni[id] = 0.0f;

    Nave[id] = 0.0f;
    for (ip = 0; ip < 3; ip++) {
      dpvdri[ip][id] = 0.0f;
      for (m = 0; m < MAX_ILP_NEIGHBOR_TMD; m++) {
        dnn[m][id] = 0.0f;
        pvet[m][id] = 0.0f;
        dpvet1[m][ip][id] = 0.0f;
        dpvet2[m][ip][id] = 0.0f;
        dNave[id][m][ip] = 0.0f;
      }
    }
  }

  if (cont <= 1) {
    normal[0] = 0.0f;
    normal[1] = 0.0f;
    normal[2] = 1.0f;
    for (id = 0; id < 3; ++id) {
      for (ip = 0; ip < 3; ++ip) {
        dnormdri[id][ip] = 0.0f;
        for (m = 0; m < MAX_ILP_NEIGHBOR_TMD; ++m) {
          dnormal[id][m][ip] = 0.0f;
        }
      }
    }
  } else if (cont > 1 && cont < MAX_ILP_NEIGHBOR_TMD) {
    for (int k = 0; k < cont - 1; ++k) {
      for (ip = 0; ip < 3; ++ip) {
        pvet[k][ip] = vect[k][modulo(ip + 1, 3)] * vect[k + 1][modulo(ip + 2, 3)] -
                vect[k][modulo(ip + 2, 3)] * vect[k + 1][modulo(ip + 1, 3)];
      }
      // dpvet1[k][l][ip]: the derivatve of the k (=0,...cont-1)th Nik respect to the ip component of atom l
      // derivatives respect to atom l
      // dNik,x/drl
      dpvet1[k][0][0] = 0.0f;
      dpvet1[k][0][1] = vect[modulo(k + 1, MAX_ILP_NEIGHBOR_TMD)][2];
      dpvet1[k][0][2] = -vect[modulo(k + 1, MAX_ILP_NEIGHBOR_TMD)][1];
      // dNik,y/drl
      dpvet1[k][1][0] = -vect[modulo(k + 1, MAX_ILP_NEIGHBOR_TMD)][2];
      dpvet1[k][1][1] = 0.0f;
      dpvet1[k][1][2] = vect[modulo(k + 1, MAX_ILP_NEIGHBOR_TMD)][0];
      // dNik,z/drl
      dpvet1[k][2][0] = vect[modulo(k + 1, MAX_ILP_NEIGHBOR_TMD)][1];
      dpvet1[k][2][1] = -vect[modulo(k + 1, MAX_ILP_NEIGHBOR_TMD)][0];
      dpvet1[k][2][2] = 0.0f;

      // dpvet2[k][l][ip]: the derivatve of the k (=0,...cont-1)th Nik respect to the ip component of atom l+1
      // derivatives respect to atom l+1
      // dNik,x/drl+1
      dpvet2[k][0][0] = 0.0f;
      dpvet2[k][0][1] = -vect[modulo(k, MAX_ILP_NEIGHBOR_TMD)][2];
      dpvet2[k][0][2] = vect[modulo(k, MAX_ILP_NEIGHBOR_TMD)][1];
      // dNik,y/drl+1
      dpvet2[k][1][0] = vect[modulo(k, MAX_ILP_NEIGHBOR_TMD)][2];
      dpvet2[k][1][1] = 0.0f;
      dpvet2[k][1][2] = -vect[modulo(k, MAX_ILP_NEIGHBOR_TMD)][0];
      // dNik,z/drl+1
      dpvet2[k][2][0] = -vect[modulo(k, MAX_ILP_NEIGHBOR_TMD)][1];
      dpvet2[k][2][1] = vect[modulo(k, MAX_ILP_NEIGHBOR_TMD)][0];
      dpvet2[k][2][2] = 0.0f;
    }

    // average the normal vectors by using the MAX_ILP_NEIGHBOR_TMD neighboring planes
    for (ip = 0; ip < 3; ip++) {
      Nave[ip] = 0.0f;
      for (int k = 0; k < cont - 1; k++) {
        Nave[ip] += pvet[k][ip];
      }
      Nave[ip] /= (cont - 1);
    }
    nninv = rnorm3df(Nave[0], Nave[1], Nave[2]);
    
    // the unit normal vector
    normal[0] = Nave[0] * nninv;
    normal[1] = Nave[1] * nninv;
    normal[2] = Nave[2] * nninv;

    // derivatives of non-normalized normal vector, dNave:3xcontx3 array
    // dNave[id][m][ip]: the derivatve of the id component of Nave respect to the ip component of atom m
    for (id = 0; id < 3; id++) {
      for (ip = 0; ip < 3; ip++) {
        for (m = 0; m < cont; m++) {
          if (m == 0) {
            dNave[id][m][ip] = dpvet1[m][id][ip] / (cont - 1);
          } else if (m == cont - 1) {
            dNave[id][m][ip] = dpvet2[m - 1][id][ip] / (cont - 1);
          } else {    // sum of the derivatives of the mth and (m-1)th normal vector respect to the atom m
            dNave[id][m][ip] = (dpvet1[m][id][ip] + dpvet2[m - 1][id][ip]) / (cont - 1);
          }
        }
      }
    }
    // derivatives of nn, dnn:contx3 vector
    // dnn[m][id]: the derivative of nn respect to r[m][id], m=0,...MAX_ILP_NEIGHBOR_TMD-1; id=0,1,2
    // r[m][id]: the id's component of atom m
    for (m = 0; m < cont; m++) {
      for (id = 0; id < 3; id++) {
        dnn[m][id] = (Nave[0] * dNave[0][m][id] + Nave[1] * dNave[1][m][id] +
                      Nave[2] * dNave[2][m][id]) * nninv;
      }
    }
    // dnormal[i][id][m][ip]: the derivative of normal[i][id] respect to r[m][ip], id,ip=0,1,2.
    // for atom m, which is a neighbor atom of atom i, m = 0,...,MAX_ILP_NEIGHBOR_TMD-1
    for (m = 0; m < cont; m++) {
      for (id = 0; id < 3; id++) {
        for (ip = 0; ip < 3; ip++) {
          dnormal[id][m][ip] = dNave[id][m][ip] * nninv - Nave[id] * dnn[m][ip] * nninv * nninv;
        }
      }
    }
    // Calculte dNave/dri, defined as dpvdri
    for (id = 0; id < 3; id++) {
      for (ip = 0; ip < 3; ip++) {
        dpvdri[id][ip] = 0.0;
        for (int k = 0; k < cont; k++) {
          dpvdri[id][ip] -= dNave[id][k][ip];
        }
      }
    }

    // derivatives of nn, dnn:3x1 vector
    dni[0] = (Nave[0] * dpvdri[0][0] + Nave[1] * dpvdri[1][0] + Nave[2] * dpvdri[2][0]) * nninv;
    dni[1] = (Nave[0] * dpvdri[0][1] + Nave[1] * dpvdri[1][1] + Nave[2] * dpvdri[2][1]) * nninv;
    dni[2] = (Nave[0] * dpvdri[0][2] + Nave[1] * dpvdri[1][2] + Nave[2] * dpvdri[2][2]) * nninv;
    // derivatives of unit vector ni respect to ri, the result is 3x3 matrix
    for (id = 0; id < 3; id++) {
      for (ip = 0; ip < 3; ip++) {
        dnormdri[id][ip] = dpvdri[id][ip] * nninv - Nave[id] * dni[ip] * nninv * nninv;
      }
    }
  } else if (cont == MAX_ILP_NEIGHBOR_TMD) {
    // derivatives of Ni[l] respect to the MAX_ILP_NEIGHBOR_TMD neighbors
    for (int k = 0; k < MAX_ILP_NEIGHBOR_TMD; ++k) {
      for (ip = 0; ip < 3; ++ip) {
        pvet[k][ip] = vect[modulo(k, MAX_ILP_NEIGHBOR_TMD)][modulo(ip + 1, 3)] *
                vect[modulo(k + 1, MAX_ILP_NEIGHBOR_TMD)][modulo(ip + 2, 3)] -
            vect[modulo(k, MAX_ILP_NEIGHBOR_TMD)][modulo(ip + 2, 3)] *
                vect[modulo(k + 1, MAX_ILP_NEIGHBOR_TMD)][modulo(ip + 1, 3)];
      }
      // dpvet1[k][l][ip]: the derivatve of the k (=0,...cont-1)th Nik respect to the ip component of atom l
      // derivatives respect to atom l
      // dNik,x/drl
      dpvet1[k][0][0] = 0.0f;
      dpvet1[k][0][1] = vect[modulo(k + 1, MAX_ILP_NEIGHBOR_TMD)][2];
      dpvet1[k][0][2] = -vect[modulo(k + 1, MAX_ILP_NEIGHBOR_TMD)][1];
      // dNik,y/drl
      dpvet1[k][1][0] = -vect[modulo(k + 1, MAX_ILP_NEIGHBOR_TMD)][2];
      dpvet1[k][1][1] = 0.0f;
      dpvet1[k][1][2] = vect[modulo(k + 1, MAX_ILP_NEIGHBOR_TMD)][0];
      // dNik,z/drl
      dpvet1[k][2][0] = vect[modulo(k + 1, MAX_ILP_NEIGHBOR_TMD)][1];
      dpvet1[k][2][1] = -vect[modulo(k + 1, MAX_ILP_NEIGHBOR_TMD)][0];
      dpvet1[k][2][2] = 0.0f;

      // dpvet2[k][l][ip]: the derivatve of the k (=0,...cont-1)th Nik respect to the ip component of atom l+1
      // derivatives respect to atom l+1
      // dNik,x/drl+1
      dpvet2[k][0][0] = 0.0f;
      dpvet2[k][0][1] = -vect[modulo(k, MAX_ILP_NEIGHBOR_TMD)][2];
      dpvet2[k][0][2] = vect[modulo(k, MAX_ILP_NEIGHBOR_TMD)][1];
      // dNik,y/drl+1
      dpvet2[k][1][0] = vect[modulo(k, MAX_ILP_NEIGHBOR_TMD)][2];
      dpvet2[k][1][1] = 0.0f;
      dpvet2[k][1][2] = -vect[modulo(k, MAX_ILP_NEIGHBOR_TMD)][0];
      // dNik,z/drl+1
      dpvet2[k][2][0] = -vect[modulo(k, MAX_ILP_NEIGHBOR_TMD)][1];
      dpvet2[k][2][1] = vect[modulo(k, MAX_ILP_NEIGHBOR_TMD)][0];
      dpvet2[k][2][2] = 0.0f;
    }

    // average the normal vectors by using the MAX_ILP_NEIGHBOR_TMD neighboring planes
    for (ip = 0; ip < 3; ++ip) {
      Nave[ip] = 0.0f;
      for (int k = 0; k < MAX_ILP_NEIGHBOR_TMD; ++k) {
        Nave[ip] += pvet[k][ip];
      }
      Nave[ip] /= MAX_ILP_NEIGHBOR_TMD;
    }
    // the magnitude of the normal vector
    // nn2 = Nave[0] * Nave[0] + Nave[1] * Nave[1] + Nave[2] * Nave[2];
    nninv = rnorm3df(Nave[0], Nave[1], Nave[2]);
    // the unit normal vector
    normal[0] = Nave[0] * nninv;
    normal[1] = Nave[1] * nninv;
    normal[2] = Nave[2] * nninv;

    // for the central atoms, dnormdri is always zero
    for (id = 0; id < 3; ++id) {
      for (ip = 0; ip < 3; ++ip) {
        dnormdri[id][ip] = 0.0f;
      }
    }

    // derivatives of non-normalized normal vector, dNave:3xMAX_ILP_NEIGHBOR_TMDx3 array
    // dNave[id][m][ip]: the derivatve of the id component of Nave respect to the ip component of atom m
    for (id = 0; id < 3; ++id) {
      for (ip = 0; ip < 3; ++ip) {
        for (
            m = 0; m < MAX_ILP_NEIGHBOR_TMD;
            ++m) {    // sum of the derivatives of the mth and (m-1)th normal vector respect to the atom m
          dNave[id][m][ip] =
              (dpvet1[modulo(m, MAX_ILP_NEIGHBOR_TMD)][id][ip] + dpvet2[modulo(m - 1, MAX_ILP_NEIGHBOR_TMD)][id][ip]) / MAX_ILP_NEIGHBOR_TMD;
        }
      }
    }
    // derivatives of nn, dnn:MAX_ILP_NEIGHBOR_TMDx3 vector
    // dnn[m][id]: the derivative of nn respect to r[m][id], m=0,...MAX_ILP_NEIGHBOR_TMD-1; id=0,1,2
    // r[m][id]: the id's component of atom m
    for (m = 0; m < MAX_ILP_NEIGHBOR_TMD; ++m) {
      for (id = 0; id < 3; ++id) {
        dnn[m][id] =
            (Nave[0] * dNave[0][m][id] + Nave[1] * dNave[1][m][id] + Nave[2] * dNave[2][m][id]) *
            nninv;
      }
    }
    // dnormal[i][id][m][ip]: the derivative of normal[i][id] respect to r[m][ip], id,ip=0,1,2.
    // for atom m, which is a neighbor atom of atom i, m = 0,...,MAX_ILP_NEIGHBOR_TMD-1
    for (m = 0; m < MAX_ILP_NEIGHBOR_TMD; ++m) {
      for (id = 0; id < 3; ++id) {
        for (ip = 0; ip < 3; ++ip) {
          dnormal[id][m][ip] = dNave[id][m][ip] * nninv - Nave[id] * dnn[m][ip] * nninv * nninv;
        }
      }
    }
  } else {
    printf("\n===== ILP neighbor number[%d] is greater than 6 =====\n", cont);
    exit(1);
  }
}

// calculate the van der Waals force and energy
static __device__ void calc_vdW(
  float r,
  float rinv,
  float rsq,
  float d,
  float d_Seff,
  float C_6,
  float Tap,
  float dTap,
  float &p2_vdW,
  float &f2_vdW)
{
  float r2inv, r6inv, r8inv;
  float TSvdw, TSvdwinv, Vilp;
  float fpair, fsum;

  r2inv = 1.0f / rsq;
  r6inv = r2inv * r2inv * r2inv;
  r8inv = r2inv * r6inv;

  // TSvdw = 1.0 + exp(-d_Seff * r + d);
  TSvdw = 1.0f + expf(-d_Seff * r + d);
  TSvdwinv = 1.0f / TSvdw;
  Vilp = -C_6 * r6inv * TSvdwinv;

  // derivatives
  // fpair = -6.0 * C_6 * r8inv * TSvdwinv + \
  //   C_6 * d_Seff * (TSvdw - 1.0) * TSvdwinv * TSvdwinv * r8inv * r;
  fpair = (-6.0f + d_Seff * (TSvdw - 1.0f) * TSvdwinv * r ) * C_6 * TSvdwinv * r8inv;
  fsum = fpair * Tap - Vilp * dTap * rinv;

  p2_vdW = Tap * Vilp;
  f2_vdW = fsum;
}

// force evaluation kernel
static __global__ void gpu_find_force(
  ILP_Para ilp_para,
  const int number_of_particles,
  const int N1,
  const int N2,
  const Box box,
  const int *g_neighbor_number,
  const int *g_neighbor_list,
  int *g_ilp_neighbor_number,
  int *g_ilp_neighbor_list,
  const int *group_label,
  const int *g_type,
  const double *__restrict__ g_x,
  const double *__restrict__ g_y,
  const double *__restrict__ g_z,
  double *g_fx,
  double *g_fy,
  double *g_fz,
  double *g_virial,
  double *g_potential,
  float *g_f12x,
  float *g_f12y,
  float *g_f12z,
  float *g_f12x_ilp_neigh,
  float *g_f12y_ilp_neigh,
  float *g_f12z_ilp_neigh,
  bool sublayer_flag[MAX_TYPE_ILP_NEP])
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1; // particle index
  float s_fx = 0.0f;                                   // force_x
  float s_fy = 0.0f;                                   // force_y
  float s_fz = 0.0f;                                   // force_z
  float s_pe = 0.0f;                                   // potential energy
  float s_sxx = 0.0f;                                  // virial_stress_xx
  float s_sxy = 0.0f;                                  // virial_stress_xy
  float s_sxz = 0.0f;                                  // virial_stress_xz
  float s_syx = 0.0f;                                  // virial_stress_yx
  float s_syy = 0.0f;                                  // virial_stress_yy
  float s_syz = 0.0f;                                  // virial_stress_yz
  float s_szx = 0.0f;                                  // virial_stress_zx
  float s_szy = 0.0f;                                  // virial_stress_zy
  float s_szz = 0.0f;                                  // virial_stress_zz

  float r = 0.0f;
  float rsq = 0.0f;
  float Rcut = 0.0f;

  if (n1 < N2) {
    double x12d, y12d, z12d;
    float x12f, y12f, z12f;
    int neighor_number = g_neighbor_number[n1];
    int type1 = g_type[n1];
    double x1 = g_x[n1];
    double y1 = g_y[n1];
    double z1 = g_z[n1];

    float delkix_half[MAX_ILP_NEIGHBOR_TMD] = {0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f};
    float delkiy_half[MAX_ILP_NEIGHBOR_TMD] = {0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f};
    float delkiz_half[MAX_ILP_NEIGHBOR_TMD] = {0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f};

    // calculate the normal
    int cont = 0;
    float normal[3];
    float dnormdri[3][3];
    float dnormal[3][MAX_ILP_NEIGHBOR_TMD][3];

    float vet[MAX_ILP_NEIGHBOR_TMD][3];
    int id, ip, m;
    for (id = 0; id < 3; ++id) {
      normal[id] = 0.0f;
      for (ip = 0; ip < 3; ++ip) {
        dnormdri[ip][id] = 0.0f;
        for (m = 0; m < MAX_ILP_NEIGHBOR_TMD; ++m) {
          dnormal[id][m][ip] = 0.0f;
          vet[m][id] = 0.0f;
        }
      }
    }

    int ilp_neighbor_number = g_ilp_neighbor_number[n1];
    for (int i1 = 0; i1 < ilp_neighbor_number; ++i1) {
      int n2_ilp = g_ilp_neighbor_list[n1 + number_of_particles * i1];
      x12d = g_x[n2_ilp] - x1;
      y12d = g_y[n2_ilp] - y1;
      z12d = g_z[n2_ilp] - z1;
      apply_mic(box, x12d, y12d, z12d);
      vet[cont][0] = float(x12d);
      vet[cont][1] = float(y12d);
      vet[cont][2] = float(z12d);
      ++cont;

      delkix_half[i1] = float(x12d) * 0.5f;
      delkiy_half[i1] = float(y12d) * 0.5f;
      delkiz_half[i1] = float(z12d) * 0.5f;
    }
    
    if (sublayer_flag[type1]) {
      calc_normal_tmd(vet, cont, normal, dnormdri, dnormal);
    } else {
      calc_normal_cbn(vet, cont, normal, dnormdri, dnormal);
    }

    // calculate energy and force
    for (int i1 = 0; i1 < neighor_number; ++i1) {
      int index = n1 + number_of_particles * i1;
      int n2 = g_neighbor_list[index];
      int type2 = g_type[n2];

      x12d = g_x[n2] - x1;
      y12d = g_y[n2] - y1;
      z12d = g_z[n2] - z1;
      apply_mic(box, x12d, y12d, z12d);

      // save x12, y12, z12 in float
      x12f = float(x12d);
      y12f = float(y12d);
      z12f = float(z12d);

      // calculate distance between atoms
      rsq = x12f * x12f + y12f * y12f + z12f * z12f;
      r = sqrtf(rsq);
      Rcut = ilp_para.rcut_global[type1][type2];

      if (r >= Rcut) {
        continue;
      }

      // calc att
      float Tap, dTap, rinv;
      float Rcutinv = 1.0f / Rcut;
      rinv = 1.0f / r;
      Tap = calc_Tap(r, Rcutinv);
      dTap = calc_dTap(r, Rcutinv);

      float p2_vdW, f2_vdW;
      calc_vdW(
        r,
        rinv,
        rsq,
        ilp_para.d[type1][type2],
        ilp_para.d_Seff[type1][type2],
        ilp_para.C_6[type1][type2],
        Tap,
        dTap,
        p2_vdW,
        f2_vdW);
      
      float f12x = -f2_vdW * x12f * 0.5f;
      float f12y = -f2_vdW * y12f * 0.5f;
      float f12z = -f2_vdW * z12f * 0.5f;
      float f21x = -f12x;
      float f21y = -f12y;
      float f21z = -f12z;

      s_fx += f12x - f21x;
      s_fy += f12y - f21y;
      s_fz += f12z - f21z;

      s_pe += p2_vdW * 0.5f;
      s_sxx += x12f * f21x;
      s_sxy += x12f * f21y;
      s_sxz += x12f * f21z;
      s_syx += y12f * f21x;
      s_syy += y12f * f21y;
      s_syz += y12f * f21z;
      s_szx += z12f * f21x;
      s_szy += z12f * f21y;
      s_szz += z12f * f21z;

      
      // calc rep
      float C = ilp_para.C[type1][type2];
      float lambda_ = ilp_para.lambda[type1][type2];
      float delta2inv = ilp_para.delta2inv[type1][type2];
      float epsilon = ilp_para.epsilon[type1][type2];
      float z0 = ilp_para.z0[type1][type2];
      // calc_rep
      float prodnorm1, rhosq1, rdsq1, exp0, exp1, frho1, Erep, Vilp;
      float fpair, fpair1, fsum, delx, dely, delz, fkcx, fkcy, fkcz;
      float dprodnorm1[3] = {0.0f, 0.0f, 0.0f};
      float fp1[3] = {0.0f, 0.0f, 0.0f};
      float fprod1[3] = {0.0f, 0.0f, 0.0f};
      float fk[3] = {0.0f, 0.0f, 0.0f};

      delx = -x12f;
      dely = -y12f;
      delz = -z12f;

      float delx_half = delx * 0.5f;
      float dely_half = dely * 0.5f;
      float delz_half = delz * 0.5f;

      // calculate the transverse distance
      prodnorm1 = normal[0] * delx + normal[1] * dely + normal[2] * delz;
      rhosq1 = rsq - prodnorm1 * prodnorm1;
      rdsq1 = rhosq1 * delta2inv;

      // store exponents
      // exp0 = exp(-lambda_ * (r - z0));
      // exp1 = exp(-rdsq1);
      exp0 = expf(-lambda_ * (r - z0));
      exp1 = expf(-rdsq1);

      frho1 = exp1 * C;
      Erep = 0.5f * epsilon + frho1;
      Vilp = exp0 * Erep;

      // derivatives
      fpair = lambda_ * exp0 * rinv * Erep;
      fpair1 = 2.0f * exp0 * frho1 * delta2inv;
      fsum = fpair + fpair1;

      float prodnorm1_m_fpair1 = prodnorm1 * fpair1;
      float Vilp_m_dTap_m_rinv = Vilp * dTap * rinv;

      // derivatives of the product of rij and ni, the resutl is a vector
      dprodnorm1[0] = 
        dnormdri[0][0] * delx + dnormdri[1][0] * dely + dnormdri[2][0] * delz;
      dprodnorm1[1] = 
        dnormdri[0][1] * delx + dnormdri[1][1] * dely + dnormdri[2][1] * delz;
      dprodnorm1[2] = 
        dnormdri[0][2] * delx + dnormdri[1][2] * dely + dnormdri[2][2] * delz;
      // fp1[0] = prodnorm1 * normal[0] * fpair1;
      // fp1[1] = prodnorm1 * normal[1] * fpair1;
      // fp1[2] = prodnorm1 * normal[2] * fpair1;
      // fprod1[0] = prodnorm1 * dprodnorm1[0] * fpair1;
      // fprod1[1] = prodnorm1 * dprodnorm1[1] * fpair1;
      // fprod1[2] = prodnorm1 * dprodnorm1[2] * fpair1;
      fp1[0] = prodnorm1_m_fpair1 * normal[0];
      fp1[1] = prodnorm1_m_fpair1 * normal[1];
      fp1[2] = prodnorm1_m_fpair1 * normal[2];
      fprod1[0] = prodnorm1_m_fpair1 * dprodnorm1[0];
      fprod1[1] = prodnorm1_m_fpair1 * dprodnorm1[1];
      fprod1[2] = prodnorm1_m_fpair1 * dprodnorm1[2];

      // fkcx = (delx * fsum - fp1[0]) * Tap - Vilp * dTap * delx * rinv;
      // fkcy = (dely * fsum - fp1[1]) * Tap - Vilp * dTap * dely * rinv;
      // fkcz = (delz * fsum - fp1[2]) * Tap - Vilp * dTap * delz * rinv;
      fkcx = (delx * fsum - fp1[0]) * Tap - Vilp_m_dTap_m_rinv * delx;
      fkcy = (dely * fsum - fp1[1]) * Tap - Vilp_m_dTap_m_rinv * dely;
      fkcz = (delz * fsum - fp1[2]) * Tap - Vilp_m_dTap_m_rinv * delz;

      s_fx += fkcx - fprod1[0] * Tap;
      s_fy += fkcy - fprod1[1] * Tap;
      s_fz += fkcz - fprod1[2] * Tap;

      g_f12x[index] = fkcx;
      g_f12y[index] = fkcy;
      g_f12z[index] = fkcz;

      float minus_prodnorm1_m_fpair1_m_Tap = -prodnorm1 * fpair1 * Tap;
      for (int kk = 0; kk < ilp_neighbor_number; ++kk) {
      // for (int kk = 0; kk < 0; ++kk) {
        // int index_ilp = n1 + number_of_particles * kk;
        // int n2_ilp = g_ilp_neighbor_list[index_ilp];
        // derivatives of the product of rij and ni respect to rk, k=0,1,2, where atom k is the neighbors of atom i
        dprodnorm1[0] = dnormal[0][kk][0] * delx + dnormal[1][kk][0] * dely +
            dnormal[2][kk][0] * delz;
        dprodnorm1[1] = dnormal[0][kk][1] * delx + dnormal[1][kk][1] * dely +
            dnormal[2][kk][1] * delz;
        dprodnorm1[2] = dnormal[0][kk][2] * delx + dnormal[1][kk][2] * dely +
            dnormal[2][kk][2] * delz;
        // fk[0] = (-prodnorm1 * dprodnorm1[0] * fpair1) * Tap;
        // fk[1] = (-prodnorm1 * dprodnorm1[1] * fpair1) * Tap;
        // fk[2] = (-prodnorm1 * dprodnorm1[2] * fpair1) * Tap;
        fk[0] = minus_prodnorm1_m_fpair1_m_Tap * dprodnorm1[0];
        fk[1] = minus_prodnorm1_m_fpair1_m_Tap * dprodnorm1[1];
        fk[2] = minus_prodnorm1_m_fpair1_m_Tap * dprodnorm1[2];

        g_f12x_ilp_neigh[n1 + number_of_particles * kk] += fk[0];
        g_f12y_ilp_neigh[n1 + number_of_particles * kk] += fk[1];
        g_f12z_ilp_neigh[n1 + number_of_particles * kk] += fk[2];

        // delki[0] = g_x[n2_ilp] - x1;
        // delki[1] = g_y[n2_ilp] - y1;
        // delki[2] = g_z[n2_ilp] - z1;
        // apply_mic(box, delki[0], delki[1], delki[2]);

        // s_sxx += delki[0] * fk[0] * 0.5;
        // s_sxy += delki[0] * fk[1] * 0.5;
        // s_sxz += delki[0] * fk[2] * 0.5;
        // s_syx += delki[1] * fk[0] * 0.5;
        // s_syy += delki[1] * fk[1] * 0.5;
        // s_syz += delki[1] * fk[2] * 0.5;
        // s_szx += delki[2] * fk[0] * 0.5;
        // s_szy += delki[2] * fk[1] * 0.5;
        // s_szz += delki[2] * fk[2] * 0.5;
        s_sxx += delkix_half[kk] * fk[0];
        s_sxy += delkix_half[kk] * fk[1];
        s_sxz += delkix_half[kk] * fk[2];
        s_syx += delkiy_half[kk] * fk[0];
        s_syy += delkiy_half[kk] * fk[1];
        s_syz += delkiy_half[kk] * fk[2];
        s_szx += delkiz_half[kk] * fk[0];
        s_szy += delkiz_half[kk] * fk[1];
        s_szz += delkiz_half[kk] * fk[2];
      }
      s_pe += Tap * Vilp;
      s_sxx += delx_half * fkcx;
      s_sxy += delx_half * fkcy;
      s_sxz += delx_half * fkcz;
      s_syx += dely_half * fkcx;
      s_syy += dely_half * fkcy;
      s_syz += dely_half * fkcz;
      s_szx += delz_half * fkcx;
      s_szy += delz_half * fkcy;
      s_szz += delz_half * fkcz;
    }

    // save force
    g_fx[n1] += s_fx;
    g_fy[n1] += s_fy;
    g_fz[n1] += s_fz;

    // save virial
    // xx xy xz    0 3 4
    // yx yy yz    6 1 5
    // zx zy zz    7 8 2
    g_virial[n1 + 0 * number_of_particles] += s_sxx;
    g_virial[n1 + 1 * number_of_particles] += s_syy;
    g_virial[n1 + 2 * number_of_particles] += s_szz;
    g_virial[n1 + 3 * number_of_particles] += s_sxy;
    g_virial[n1 + 4 * number_of_particles] += s_sxz;
    g_virial[n1 + 5 * number_of_particles] += s_syz;
    g_virial[n1 + 6 * number_of_particles] += s_syx;
    g_virial[n1 + 7 * number_of_particles] += s_szx;
    g_virial[n1 + 8 * number_of_particles] += s_szy;

    // save potential
    g_potential[n1] += s_pe;

  }
}

// build a neighbor list for reducing force
static __global__ void build_reduce_neighbor_list(
  const int number_of_particles,
  const int N1,
  const int N2,
  const int *g_neighbor_number,
  const int *g_neighbor_list,
  int *g_reduce_neighbor_list)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1;
  if (N1 < N2) {
    int neighbor_number = g_neighbor_number[n1];
    int l, r, m, tmp_value;
    for (int i1 = 0; i1 < neighbor_number; ++i1) {
      int index = n1 + i1 * number_of_particles;
      int n2 = g_neighbor_list[index];

      l = 0;
      r = g_neighbor_number[n2];
      while (l < r) {
        m = (l + r) >> 1;
        tmp_value = g_neighbor_list[n2 + number_of_particles * m];
        if (tmp_value < n1) {
          l = m + 1;
        } else if (tmp_value > n1) {
          r = m - 1;
        } else {
          break;
        }
      }
      g_reduce_neighbor_list[index] = (l + r) >> 1;
    }
  }
}

// reduce the rep force
static __global__ void reduce_force_many_body(
  const int number_of_particles,
  const int N1,
  const int N2,
  const Box box,
  const int *g_neighbor_number,
  const int *g_neighbor_list,
  int *g_reduce_neighbor_list,
  int *g_ilp_neighbor_number,
  int *g_ilp_neighbor_list,
  const double *__restrict__ g_x,
  const double *__restrict__ g_y,
  const double *__restrict__ g_z,
  double *g_fx,
  double *g_fy,
  double *g_fz,
  double *g_virial,
  float *g_f12x,
  float *g_f12y,
  float *g_f12z,
  float *g_f12x_ilp_neigh,
  float *g_f12y_ilp_neigh,
  float *g_f12z_ilp_neigh)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1; // particle index
  float s_fx = 0.0f;                                   // force_x
  float s_fy = 0.0f;                                   // force_y
  float s_fz = 0.0f;                                   // force_z
  float s_sxx = 0.0f;                                  // virial_stress_xx
  float s_sxy = 0.0f;                                  // virial_stress_xy
  float s_sxz = 0.0f;                                  // virial_stress_xz
  float s_syx = 0.0f;                                  // virial_stress_yx
  float s_syy = 0.0f;                                  // virial_stress_yy
  float s_syz = 0.0f;                                  // virial_stress_yz
  float s_szx = 0.0f;                                  // virial_stress_zx
  float s_szy = 0.0f;                                  // virial_stress_zy
  float s_szz = 0.0f;                                  // virial_stress_zz


  if (n1 < N2) {
    double x12d, y12d, z12d;
    float x12f, y12f, z12f;
    int neighbor_number_1 = g_neighbor_number[n1];
    double x1 = g_x[n1];
    double y1 = g_y[n1];
    double z1 = g_z[n1];

    // calculate energy and force
    for (int i1 = 0; i1 < neighbor_number_1; ++i1) {
      int index = n1 + number_of_particles * i1;
      int n2 = g_neighbor_list[index];

      x12d = g_x[n2] - x1;
      y12d = g_y[n2] - y1;
      z12d = g_z[n2] - z1;
      apply_mic(box, x12d, y12d, z12d);
      x12f = float(x12d);
      y12f = float(y12d);
      z12f = float(z12d);

      index = n2 + number_of_particles * g_reduce_neighbor_list[index];
      float f21x = g_f12x[index];
      float f21y = g_f12y[index];
      float f21z = g_f12z[index];

      s_fx -= f21x;
      s_fy -= f21y;
      s_fz -= f21z;

      // per-atom virial
      s_sxx += x12f * f21x * 0.5f;
      s_sxy += x12f * f21y * 0.5f;
      s_sxz += x12f * f21z * 0.5f;
      s_syx += y12f * f21x * 0.5f;
      s_syy += y12f * f21y * 0.5f;
      s_syz += y12f * f21z * 0.5f;
      s_szx += z12f * f21x * 0.5f;
      s_szy += z12f * f21y * 0.5f;
      s_szz += z12f * f21z * 0.5f;
    }

    int ilp_neighbor_number_1 = g_ilp_neighbor_number[n1];

    for (int i1 = 0; i1 < ilp_neighbor_number_1; ++i1) {
      int index = n1 + number_of_particles * i1;
      int n2 = g_ilp_neighbor_list[index];
      int ilp_neighor_number_2 = g_ilp_neighbor_number[n2];

      x12d = g_x[n2] - x1;
      y12d = g_y[n2] - y1;
      z12d = g_z[n2] - z1;
      apply_mic(box, x12d, y12d, z12d);
      x12f = float(x12d);
      y12f = float(y12d);
      z12f = float(z12d);

      int offset = 0;
      for (int k = 0; k < ilp_neighor_number_2; ++k) {
        if (n1 == g_ilp_neighbor_list[n2 + number_of_particles * k]) {
          offset = k;
          break;
        }
      }
      index = n2 + number_of_particles * offset;
      float f21x = g_f12x_ilp_neigh[index];
      float f21y = g_f12y_ilp_neigh[index];
      float f21z = g_f12z_ilp_neigh[index];

      s_fx += f21x;
      s_fy += f21y;
      s_fz += f21z;

      // per-atom virial
      s_sxx += -x12f * f21x * 0.5f;
      s_sxy += -x12f * f21y * 0.5f;
      s_sxz += -x12f * f21z * 0.5f;
      s_syx += -y12f * f21x * 0.5f;
      s_syy += -y12f * f21y * 0.5f;
      s_syz += -y12f * f21z * 0.5f;
      s_szx += -z12f * f21x * 0.5f;
      s_szy += -z12f * f21y * 0.5f;
      s_szz += -z12f * f21z * 0.5f;
    }

    // save force
    g_fx[n1] += s_fx;
    g_fy[n1] += s_fy;
    g_fz[n1] += s_fz;

    // save virial
    // xx xy xz    0 3 4
    // yx yy yz    6 1 5
    // zx zy zz    7 8 2
    g_virial[n1 + 0 * number_of_particles] += s_sxx;
    g_virial[n1 + 1 * number_of_particles] += s_syy;
    g_virial[n1 + 2 * number_of_particles] += s_szz;
    g_virial[n1 + 3 * number_of_particles] += s_sxy;
    g_virial[n1 + 4 * number_of_particles] += s_sxz;
    g_virial[n1 + 5 * number_of_particles] += s_syz;
    g_virial[n1 + 6 * number_of_particles] += s_syx;
    g_virial[n1 + 7 * number_of_particles] += s_szx;
    g_virial[n1 + 8 * number_of_particles] += s_szy;
  }
}



// ----- NEP part -----
// nep find neighbor list
static __global__ void find_neighbor_list_large_box(
  ILP_NEP::ParaMB paramb,
  const int N,
  const int N1,
  const int N2,
  const int nx,
  const int ny,
  const int nz,
  const Box box,
  const int* g_type,
  const int* __restrict__ g_cell_count,
  const int* __restrict__ g_cell_count_sum,
  const int* __restrict__ g_cell_contents,
  const double* __restrict__ g_x,
  const double* __restrict__ g_y,
  const double* __restrict__ g_z,
  int* g_NN_radial,
  int* g_NL_radial,
  int* g_NN_angular,
  int* g_NL_angular)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1;
  if (n1 >= N2) {
    return;
  }

  double x1 = g_x[n1];
  double y1 = g_y[n1];
  double z1 = g_z[n1];
  int t1 = g_type[n1];
  int count_radial = 0;
  int count_angular = 0;

  int cell_id;
  int cell_id_x;
  int cell_id_y;
  int cell_id_z;
  find_cell_id(
    box,
    x1,
    y1,
    z1,
    2.0f * paramb.rcinv_radial,
    nx,
    ny,
    nz,
    cell_id_x,
    cell_id_y,
    cell_id_z,
    cell_id);

  const int z_lim = box.pbc_z ? 2 : 0;
  const int y_lim = box.pbc_y ? 2 : 0;
  const int x_lim = box.pbc_x ? 2 : 0;

  for (int zz = -z_lim; zz <= z_lim; ++zz) {
    for (int yy = -y_lim; yy <= y_lim; ++yy) {
      for (int xx = -x_lim; xx <= x_lim; ++xx) {
        int neighbor_cell = cell_id + zz * nx * ny + yy * nx + xx;
        if (cell_id_x + xx < 0)
          neighbor_cell += nx;
        if (cell_id_x + xx >= nx)
          neighbor_cell -= nx;
        if (cell_id_y + yy < 0)
          neighbor_cell += ny * nx;
        if (cell_id_y + yy >= ny)
          neighbor_cell -= ny * nx;
        if (cell_id_z + zz < 0)
          neighbor_cell += nz * ny * nx;
        if (cell_id_z + zz >= nz)
          neighbor_cell -= nz * ny * nx;

        const int num_atoms_neighbor_cell = g_cell_count[neighbor_cell];
        const int num_atoms_previous_cells = g_cell_count_sum[neighbor_cell];

        for (int m = 0; m < num_atoms_neighbor_cell; ++m) {
          const int n2 = g_cell_contents[num_atoms_previous_cells + m];

          if (n2 < N1 || n2 >= N2 || n1 == n2) {
            continue;
          }

          double x12double = g_x[n2] - x1;
          double y12double = g_y[n2] - y1;
          double z12double = g_z[n2] - z1;
          apply_mic(box, x12double, y12double, z12double);
          float x12 = float(x12double), y12 = float(y12double), z12 = float(z12double);
          float d12_square = x12 * x12 + y12 * y12 + z12 * z12;

          int t2 = g_type[n2];
          float rc_radial = paramb.rc_radial;
          float rc_angular = paramb.rc_angular;
          if (paramb.use_typewise_cutoff) {
            int z1 = paramb.atomic_numbers[t1];
            int z2 = paramb.atomic_numbers[t2];
            rc_radial = min(
              (COVALENT_RADIUS[z1] + COVALENT_RADIUS[z2]) * paramb.typewise_cutoff_radial_factor,
              rc_radial);
            rc_angular = min(
              (COVALENT_RADIUS[z1] + COVALENT_RADIUS[z2]) * paramb.typewise_cutoff_angular_factor,
              rc_angular);
          }

          if (d12_square >= rc_radial * rc_radial) {
            continue;
          }

          g_NL_radial[count_radial++ * N + n1] = n2;

          if (d12_square < rc_angular * rc_angular) {
            g_NL_angular[count_angular++ * N + n1] = n2;
          }
        }
      }
    }
  }

  g_NN_radial[n1] = count_radial;
  g_NN_angular[n1] = count_angular;
}

static __global__ void find_descriptor(
  ILP_NEP::ParaMB paramb,
  ILP_NEP::ANN annmb,
  const int N,
  const int N1,
  const int N2,
  const Box box,
  const int* g_NN,
  const int* g_NL,
  const int* g_NN_angular,
  const int* g_NL_angular,
  const int* __restrict__ g_type,
  const double* __restrict__ g_x,
  const double* __restrict__ g_y,
  const double* __restrict__ g_z,
  const bool is_polarizability,
#ifdef USE_TABLE
  const float* __restrict__ g_gn_radial,
  const float* __restrict__ g_gn_angular,
#endif
  double* g_pe,
  float* g_Fp,
  double* g_virial,
  float* g_sum_fxyz)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1;
  if (n1 < N2) {
    int t1 = g_type[n1];
    double x1 = g_x[n1];
    double y1 = g_y[n1];
    double z1 = g_z[n1];
    float q[MAX_DIM] = {0.0f};

    // get radial descriptors
    for (int i1 = 0; i1 < g_NN[n1]; ++i1) {
      int n2 = g_NL[n1 + N * i1];
      double x12double = g_x[n2] - x1;
      double y12double = g_y[n2] - y1;
      double z12double = g_z[n2] - z1;
      apply_mic(box, x12double, y12double, z12double);
      float x12 = float(x12double), y12 = float(y12double), z12 = float(z12double);
      float d12 = sqrt(x12 * x12 + y12 * y12 + z12 * z12);

#ifdef USE_TABLE
      int index_left, index_right;
      float weight_left, weight_right;
      find_index_and_weight(
        d12 * paramb.rcinv_radial, index_left, index_right, weight_left, weight_right);
      int t12 = t1 * paramb.num_types + g_type[n2];
      for (int n = 0; n <= paramb.n_max_radial; ++n) {
        q[n] +=
          g_gn_radial[(index_left * paramb.num_types_sq + t12) * (paramb.n_max_radial + 1) + n] *
            weight_left +
          g_gn_radial[(index_right * paramb.num_types_sq + t12) * (paramb.n_max_radial + 1) + n] *
            weight_right;
      }
#else
      float fc12;
      int t2 = g_type[n2];
      float rc = paramb.rc_radial;
      if (paramb.use_typewise_cutoff) {
        rc = min(
          (COVALENT_RADIUS[paramb.atomic_numbers[t1]] +
           COVALENT_RADIUS[paramb.atomic_numbers[t2]]) *
            paramb.typewise_cutoff_radial_factor,
          rc);
      }
      float rcinv = 1.0f / rc;
      find_fc(rc, rcinv, d12, fc12);
      float fn12[MAX_NUM_N];

      find_fn(paramb.basis_size_radial, rcinv, d12, fc12, fn12);
      for (int n = 0; n <= paramb.n_max_radial; ++n) {
        float gn12 = 0.0f;
        for (int k = 0; k <= paramb.basis_size_radial; ++k) {
          int c_index = (n * (paramb.basis_size_radial + 1) + k) * paramb.num_types_sq;
          c_index += t1 * paramb.num_types + t2;
          gn12 += fn12[k] * annmb.c[c_index];
        }
        q[n] += gn12;
      }
#endif
    }

    // get angular descriptors
    for (int n = 0; n <= paramb.n_max_angular; ++n) {
      float s[NUM_OF_ABC] = {0.0f};
      for (int i1 = 0; i1 < g_NN_angular[n1]; ++i1) {
        int n2 = g_NL_angular[n1 + N * i1];
        double x12double = g_x[n2] - x1;
        double y12double = g_y[n2] - y1;
        double z12double = g_z[n2] - z1;
        apply_mic(box, x12double, y12double, z12double);
        float x12 = float(x12double), y12 = float(y12double), z12 = float(z12double);
        float d12 = sqrt(x12 * x12 + y12 * y12 + z12 * z12);
#ifdef USE_TABLE
        int index_left, index_right;
        float weight_left, weight_right;
        find_index_and_weight(
          d12 * paramb.rcinv_angular, index_left, index_right, weight_left, weight_right);
        int t12 = t1 * paramb.num_types + g_type[n2];
        float gn12 =
          g_gn_angular[(index_left * paramb.num_types_sq + t12) * (paramb.n_max_angular + 1) + n] *
            weight_left +
          g_gn_angular[(index_right * paramb.num_types_sq + t12) * (paramb.n_max_angular + 1) + n] *
            weight_right;
        accumulate_s(paramb.L_max, d12, x12, y12, z12, gn12, s);
#else
        float fc12;
        int t2 = g_type[n2];
        float rc = paramb.rc_angular;
        if (paramb.use_typewise_cutoff) {
          rc = min(
            (COVALENT_RADIUS[paramb.atomic_numbers[t1]] +
             COVALENT_RADIUS[paramb.atomic_numbers[t2]]) *
              paramb.typewise_cutoff_angular_factor,
            rc);
        }
        float rcinv = 1.0f / rc;
        find_fc(rc, rcinv, d12, fc12);
        float fn12[MAX_NUM_N];
        find_fn(paramb.basis_size_angular, rcinv, d12, fc12, fn12);
        float gn12 = 0.0f;
        for (int k = 0; k <= paramb.basis_size_angular; ++k) {
          int c_index = (n * (paramb.basis_size_angular + 1) + k) * paramb.num_types_sq;
          c_index += t1 * paramb.num_types + t2 + paramb.num_c_radial;
          gn12 += fn12[k] * annmb.c[c_index];
        }
        accumulate_s(paramb.L_max, d12, x12, y12, z12, gn12, s);
#endif
      }
      find_q(paramb.L_max, paramb.num_L, paramb.n_max_angular + 1, n, s, q + (paramb.n_max_radial + 1));
      for (int abc = 0; abc < NUM_OF_ABC; ++abc) {
        g_sum_fxyz[(n * NUM_OF_ABC + abc) * N + n1] = s[abc];
      }
    }

    // nomalize descriptor
    for (int d = 0; d < annmb.dim; ++d) {
      q[d] = q[d] * paramb.q_scaler[d];
    }

    // get energy and energy gradient
    float F = 0.0f, Fp[MAX_DIM] = {0.0f};

    if (is_polarizability) {
      apply_ann_one_layer(
        annmb.dim,
        annmb.num_neurons1,
        annmb.w0_pol[t1],
        annmb.b0_pol[t1],
        annmb.w1_pol[t1],
        annmb.b1_pol,
        q,
        F,
        Fp);
      // Add the potential F for this atom to the diagonal of the virial
      g_virial[n1] = F;
      g_virial[n1 + N * 1] = F;
      g_virial[n1 + N * 2] = F;

      // Reset the potential and forces such that they
      // are zero for the next call to the model. The next call
      // is not used in the case of is_pol = True, but it doesn't
      // hurt to clean up.
      F = 0.0f;
      for (int d = 0; d < annmb.dim; ++d) {
        Fp[d] = 0.0f;
      }
    }

    if (paramb.version == 5) {
      apply_ann_one_layer_nep5(
        annmb.dim,
        annmb.num_neurons1,
        annmb.w0[t1],
        annmb.b0[t1],
        annmb.w1[t1],
        annmb.b1,
        q,
        F,
        Fp);
    } else {
      apply_ann_one_layer(
        annmb.dim,
        annmb.num_neurons1,
        annmb.w0[t1],
        annmb.b0[t1],
        annmb.w1[t1],
        annmb.b1,
        q,
        F,
        Fp);
    }
    g_pe[n1] += F;

    for (int d = 0; d < annmb.dim; ++d) {
      g_Fp[d * N + n1] = Fp[d] * paramb.q_scaler[d];
    }
  }
}

static __global__ void find_force_radial(
  ILP_NEP::ParaMB paramb,
  ILP_NEP::ANN annmb,
  const int N,
  const int N1,
  const int N2,
  const Box box,
  const int* g_NN,
  const int* g_NL,
  const int* __restrict__ g_type,
  const double* __restrict__ g_x,
  const double* __restrict__ g_y,
  const double* __restrict__ g_z,
  const float* __restrict__ g_Fp,
  const bool is_dipole,
#ifdef USE_TABLE
  const float* __restrict__ g_gnp_radial,
#endif
  double* g_fx,
  double* g_fy,
  double* g_fz,
  double* g_virial)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1;
  if (n1 < N2) {
    int t1 = g_type[n1];
    float s_fx = 0.0f;
    float s_fy = 0.0f;
    float s_fz = 0.0f;
    float s_sxx = 0.0f;
    float s_sxy = 0.0f;
    float s_sxz = 0.0f;
    float s_syx = 0.0f;
    float s_syy = 0.0f;
    float s_syz = 0.0f;
    float s_szx = 0.0f;
    float s_szy = 0.0f;
    float s_szz = 0.0f;
    double x1 = g_x[n1];
    double y1 = g_y[n1];
    double z1 = g_z[n1];
    for (int i1 = 0; i1 < g_NN[n1]; ++i1) {
      int n2 = g_NL[n1 + N * i1];
      int t2 = g_type[n2];
      double x12double = g_x[n2] - x1;
      double y12double = g_y[n2] - y1;
      double z12double = g_z[n2] - z1;
      apply_mic(box, x12double, y12double, z12double);
      float r12[3] = {float(x12double), float(y12double), float(z12double)};
      float d12 = sqrt(r12[0] * r12[0] + r12[1] * r12[1] + r12[2] * r12[2]);
      float d12inv = 1.0f / d12;
      float f12[3] = {0.0f};
      float f21[3] = {0.0f};
#ifdef USE_TABLE
      int index_left, index_right;
      float weight_left, weight_right;
      find_index_and_weight(
        d12 * paramb.rcinv_radial, index_left, index_right, weight_left, weight_right);
      int t12 = t1 * paramb.num_types + t2;
      int t21 = t2 * paramb.num_types + t1;
      for (int n = 0; n <= paramb.n_max_radial; ++n) {
        float gnp12 =
          g_gnp_radial[(index_left * paramb.num_types_sq + t12) * (paramb.n_max_radial + 1) + n] *
            weight_left +
          g_gnp_radial[(index_right * paramb.num_types_sq + t12) * (paramb.n_max_radial + 1) + n] *
            weight_right;
        float gnp21 =
          g_gnp_radial[(index_left * paramb.num_types_sq + t21) * (paramb.n_max_radial + 1) + n] *
            weight_left +
          g_gnp_radial[(index_right * paramb.num_types_sq + t21) * (paramb.n_max_radial + 1) + n] *
            weight_right;
        float tmp12 = g_Fp[n1 + n * N] * gnp12 * d12inv;
        float tmp21 = g_Fp[n2 + n * N] * gnp21 * d12inv;
        for (int d = 0; d < 3; ++d) {
          f12[d] += tmp12 * r12[d];
          f21[d] -= tmp21 * r12[d];
        }
      }
#else
      float fc12, fcp12;
      float rc = paramb.rc_radial;
      if (paramb.use_typewise_cutoff) {
        rc = min(
          (COVALENT_RADIUS[paramb.atomic_numbers[t1]] +
           COVALENT_RADIUS[paramb.atomic_numbers[t2]]) *
            paramb.typewise_cutoff_radial_factor,
          rc);
      }
      float rcinv = 1.0f / rc;
      find_fc_and_fcp(rc, rcinv, d12, fc12, fcp12);
      float fn12[MAX_NUM_N];
      float fnp12[MAX_NUM_N];
      find_fn_and_fnp(paramb.basis_size_radial, rcinv, d12, fc12, fcp12, fn12, fnp12);
      for (int n = 0; n <= paramb.n_max_radial; ++n) {
        float gnp12 = 0.0f;
        float gnp21 = 0.0f;
        for (int k = 0; k <= paramb.basis_size_radial; ++k) {
          int c_index = (n * (paramb.basis_size_radial + 1) + k) * paramb.num_types_sq;
          gnp12 += fnp12[k] * annmb.c[c_index + t1 * paramb.num_types + t2];
          gnp21 += fnp12[k] * annmb.c[c_index + t2 * paramb.num_types + t1];
        }
        float tmp12 = g_Fp[n1 + n * N] * gnp12 * d12inv;
        float tmp21 = g_Fp[n2 + n * N] * gnp21 * d12inv;
        for (int d = 0; d < 3; ++d) {
          f12[d] += tmp12 * r12[d];
          f21[d] -= tmp21 * r12[d];
        }
      }
#endif
      s_fx += f12[0] - f21[0];
      s_fy += f12[1] - f21[1];
      s_fz += f12[2] - f21[2];
      if (is_dipole) {
        // The dipole is proportional to minus the sum of the virials times r12
        double r12_square = r12[0] * r12[0] + r12[1] * r12[1] + r12[2] * r12[2];
        s_sxx -= r12_square * f21[0];
        s_syy -= r12_square * f21[1];
        s_szz -= r12_square * f21[2];
      } else {
        s_sxx += r12[0] * f21[0];
        s_syy += r12[1] * f21[1];
        s_szz += r12[2] * f21[2];
      }
      s_sxy += r12[0] * f21[1];
      s_sxz += r12[0] * f21[2];
      s_syx += r12[1] * f21[0];
      s_syz += r12[1] * f21[2];
      s_szx += r12[2] * f21[0];
      s_szy += r12[2] * f21[1];
    }
    g_fx[n1] += s_fx;
    g_fy[n1] += s_fy;
    g_fz[n1] += s_fz;
    // save virial
    // xx xy xz    0 3 4
    // yx yy yz    6 1 5
    // zx zy zz    7 8 2
    g_virial[n1 + 0 * N] += s_sxx;
    g_virial[n1 + 1 * N] += s_syy;
    g_virial[n1 + 2 * N] += s_szz;
    g_virial[n1 + 3 * N] += s_sxy;
    g_virial[n1 + 4 * N] += s_sxz;
    g_virial[n1 + 5 * N] += s_syz;
    g_virial[n1 + 6 * N] += s_syx;
    g_virial[n1 + 7 * N] += s_szx;
    g_virial[n1 + 8 * N] += s_szy;
  }
}

static __global__ void find_partial_force_angular(
  ILP_NEP::ParaMB paramb,
  ILP_NEP::ANN annmb,
  const int N,
  const int N1,
  const int N2,
  const Box box,
  const int* g_NN_angular,
  const int* g_NL_angular,
  const int* __restrict__ g_type,
  const double* __restrict__ g_x,
  const double* __restrict__ g_y,
  const double* __restrict__ g_z,
  const float* __restrict__ g_Fp,
  const float* __restrict__ g_sum_fxyz,
#ifdef USE_TABLE
  const float* __restrict__ g_gn_angular,
  const float* __restrict__ g_gnp_angular,
#endif
  float* g_f12x,
  float* g_f12y,
  float* g_f12z)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1;
  if (n1 < N2) {

    float Fp[MAX_DIM_ANGULAR] = {0.0f};
    float sum_fxyz[NUM_OF_ABC * MAX_NUM_N];
    for (int d = 0; d < paramb.dim_angular; ++d) {
      Fp[d] = g_Fp[(paramb.n_max_radial + 1 + d) * N + n1];
    }
    for (int d = 0; d < (paramb.n_max_angular + 1) * NUM_OF_ABC; ++d) {
      sum_fxyz[d] = g_sum_fxyz[d * N + n1];
    }

    int t1 = g_type[n1];
    double x1 = g_x[n1];
    double y1 = g_y[n1];
    double z1 = g_z[n1];
    for (int i1 = 0; i1 < g_NN_angular[n1]; ++i1) {
      int index = i1 * N + n1;
      int n2 = g_NL_angular[n1 + N * i1];
      double x12double = g_x[n2] - x1;
      double y12double = g_y[n2] - y1;
      double z12double = g_z[n2] - z1;
      apply_mic(box, x12double, y12double, z12double);
      float r12[3] = {float(x12double), float(y12double), float(z12double)};
      float d12 = sqrt(r12[0] * r12[0] + r12[1] * r12[1] + r12[2] * r12[2]);
      float f12[3] = {0.0f};
#ifdef USE_TABLE
      int index_left, index_right;
      float weight_left, weight_right;
      find_index_and_weight(
        d12 * paramb.rcinv_angular, index_left, index_right, weight_left, weight_right);
      int t12 = t1 * paramb.num_types + g_type[n2];
      for (int n = 0; n <= paramb.n_max_angular; ++n) {
        int index_left_all =
          (index_left * paramb.num_types_sq + t12) * (paramb.n_max_angular + 1) + n;
        int index_right_all =
          (index_right * paramb.num_types_sq + t12) * (paramb.n_max_angular + 1) + n;
        float gn12 =
          g_gn_angular[index_left_all] * weight_left + g_gn_angular[index_right_all] * weight_right;
        float gnp12 = g_gnp_angular[index_left_all] * weight_left +
                      g_gnp_angular[index_right_all] * weight_right;
        accumulate_f12(paramb.L_max, paramb.num_L, n, paramb.n_max_angular + 1, d12, r12, gn12, gnp12, Fp, sum_fxyz, f12);
      }
#else
      float fc12, fcp12;
      int t2 = g_type[n2];
      float rc = paramb.rc_angular;
      if (paramb.use_typewise_cutoff) {
        rc = min(
          (COVALENT_RADIUS[paramb.atomic_numbers[t1]] +
           COVALENT_RADIUS[paramb.atomic_numbers[t2]]) *
            paramb.typewise_cutoff_angular_factor,
          rc);
      }
      float rcinv = 1.0f / rc;
      find_fc_and_fcp(rc, rcinv, d12, fc12, fcp12);

      float fn12[MAX_NUM_N];
      float fnp12[MAX_NUM_N];
      find_fn_and_fnp(paramb.basis_size_angular, rcinv, d12, fc12, fcp12, fn12, fnp12);
      for (int n = 0; n <= paramb.n_max_angular; ++n) {
        float gn12 = 0.0f;
        float gnp12 = 0.0f;
        for (int k = 0; k <= paramb.basis_size_angular; ++k) {
          int c_index = (n * (paramb.basis_size_angular + 1) + k) * paramb.num_types_sq;
          c_index += t1 * paramb.num_types + t2 + paramb.num_c_radial;
          gn12 += fn12[k] * annmb.c[c_index];
          gnp12 += fnp12[k] * annmb.c[c_index];
        }
        accumulate_f12(paramb.L_max, paramb.num_L, n, paramb.n_max_angular + 1, d12, r12, gn12, gnp12, Fp, sum_fxyz, f12);
      }
#endif
      g_f12x[index] = f12[0];
      g_f12y[index] = f12[1];
      g_f12z[index] = f12[2];
    }
  }
}









