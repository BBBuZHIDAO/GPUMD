#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan and GPUMD development team
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/

/*----------------------------------------------------------------------------80
The class dealing with the interlayer potential(ILP) and SW.
TODO:
------------------------------------------------------------------------------*/

#include "ilp_nep_gr_hbn.cuh"
#include "neighbor.cuh"
#include "utilities/error.cuh"
#include "utilities/common.cuh"
#include "utilities/gpu_macro.cuh"
#include "utilities/nep_utilities.cuh"
#include <fstream>
#include <iostream>
#include <string>
#include <vector>

#define BLOCK_SIZE_FORCE 128

// there are most 3 intra-layer neighbors for graphene and h-BN
#define NNEI 3

const std::string ELEMENTS[NUM_ELEMENTS] = {
  "H",  "He", "Li", "Be", "B",  "C",  "N",  "O",  "F",  "Ne", "Na", "Mg", "Al", "Si", "P",  "S",
  "Cl", "Ar", "K",  "Ca", "Sc", "Ti", "V",  "Cr", "Mn", "Fe", "Co", "Ni", "Cu", "Zn", "Ga", "Ge",
  "As", "Se", "Br", "Kr", "Rb", "Sr", "Y",  "Zr", "Nb", "Mo", "Tc", "Ru", "Rh", "Pd", "Ag", "Cd",
  "In", "Sn", "Sb", "Te", "I",  "Xe", "Cs", "Ba", "La", "Ce", "Pr", "Nd", "Pm", "Sm", "Eu", "Gd",
  "Tb", "Dy", "Ho", "Er", "Tm", "Yb", "Lu", "Hf", "Ta", "W",  "Re", "Os", "Ir", "Pt", "Au", "Hg",
  "Tl", "Pb", "Bi", "Po", "At", "Rn", "Fr", "Ra", "Ac", "Th", "Pa", "U",  "Np", "Pu"};


ILP_NEP_GR_HBN::ILP_NEP_GR_HBN(FILE* fid_ilp, const char* file_nep, int num_types, int num_atoms)
{
  // read ILP TMD potential parameter
  printf("Use %d-element ILP potential with elements:\n", num_types);
  if (!(num_types >= 1 && num_types <= MAX_TYPE_ILP_GR_HBN)) {
    PRINT_INPUT_ERROR("Incorrect type number of ILP_NEP_GR_HBN parameters.\n");
  }
  for (int n = 0; n < num_types; ++n) {
    char atom_symbol[10];
    int count = fscanf(fid_ilp, "%s", atom_symbol);
    PRINT_SCANF_ERROR(count, 1, "Reading error for ILP_NEP_GR_HBN potential.");
    printf(" %s", atom_symbol);
  }
  printf("\n");

  // read parameters
  float beta, alpha, delta, epsilon, C, d, sR;
  float reff, C6, S, rcut_ilp, rcut_global;
  rc = 0.0;
  for (int n = 0; n < num_types; ++n) {
    for (int m = 0; m < num_types; ++m) {
      int count = fscanf(fid_ilp, "%f%f%f%f%f%f%f%f%f%f%f%f", \
      &beta, &alpha, &delta, &epsilon, &C, &d, &sR, &reff, &C6, &S, \
      &rcut_ilp, &rcut_global);
      PRINT_SCANF_ERROR(count, 12, "Reading error for ILP_NEP_GR_HBN potential.");

      ilp_para.C[n][m] = C;
      ilp_para.C_6[n][m] = C6;
      ilp_para.d[n][m] = d;
      ilp_para.d_Seff[n][m] = d / sR / reff;
      ilp_para.epsilon[n][m] = epsilon;
      ilp_para.z0[n][m] = beta;
      ilp_para.lambda[n][m] = alpha / beta;
      ilp_para.delta2inv[n][m] = 1.0 / (delta * delta);
      ilp_para.S[n][m] = S;
      ilp_para.rcutsq_ilp[n][m] = rcut_ilp * rcut_ilp;
      ilp_para.rcut_global[n][m] = rcut_global;
      float meV = 1e-3 * S;
      ilp_para.C[n][m] *= meV;
      ilp_para.C_6[n][m] *= meV;
      ilp_para.epsilon[n][m] *= meV;

      if (rc < rcut_global)
        rc = rcut_global;
    }
  }


  // initialize neighbor lists and some temp vectors
  int max_neighbor_number = min(num_atoms, CUDA_MAX_NL_GR_HBN);
  ilp_data.NN.resize(num_atoms);
  ilp_data.NL.resize(num_atoms * max_neighbor_number);
  ilp_data.cell_count.resize(num_atoms);
  ilp_data.cell_count_sum.resize(num_atoms);
  ilp_data.cell_contents.resize(num_atoms);

  // init ilp neighbor list
  ilp_data.ilp_NN.resize(num_atoms);
  ilp_data.ilp_NL.resize(num_atoms * MAX_ILP_NEIGHBOR_GR_HBN);
  ilp_data.reduce_NL.resize(num_atoms * max_neighbor_number);
  ilp_data.big_ilp_NN.resize(num_atoms);
  ilp_data.big_ilp_NL.resize(num_atoms * MAX_BIG_ILP_NEIGHBOR_GR_HBN);

  ilp_data.f12x.resize(num_atoms * max_neighbor_number);
  ilp_data.f12y.resize(num_atoms * max_neighbor_number);
  ilp_data.f12z.resize(num_atoms * max_neighbor_number);

  ilp_data.f12x_ilp_neigh.resize(num_atoms * MAX_ILP_NEIGHBOR_GR_HBN);
  ilp_data.f12y_ilp_neigh.resize(num_atoms * MAX_ILP_NEIGHBOR_GR_HBN);
  ilp_data.f12z_ilp_neigh.resize(num_atoms * MAX_ILP_NEIGHBOR_GR_HBN);


  // init constant cutoff coeff
  float h_tap_coeff[8] = \
    {1.0f, 0.0f, 0.0f, 0.0f, -35.0f, 84.0f, -70.0f, 20.0f};
  CHECK(gpuMemcpyToSymbol(Tap_coeff_tmd, h_tap_coeff, 8 * sizeof(float)));

  // set ilp_flag to 1
  ilp_flag = 1;


  std::ifstream input(file_nep);
  if (!input.is_open()) {
    std::cout << "Failed to open " << file_nep << std::endl;
    exit(1);
  }

  // nep3 1 C
  std::vector<std::string> tokens = get_tokens(input);
  if (tokens.size() < 3) {
    std::cout << "The first line of nep.txt should have at least 3 items." << std::endl;
    exit(1);
  }
  if (tokens[0] == "nep3") {
    paramb.version = 3;
  } else if (tokens[0] == "nep4") {
    paramb.version = 4;
  } else if (tokens[0] == "nep5") {
    paramb.version = 5;
  } else {
    std::cout << tokens[0]
              << " is an unsupported NEP model. We only support NEP3 and NEP4 models now."
              << std::endl;
    exit(1);
  }
  paramb.num_types = get_int_from_token(tokens[1], __FILE__, __LINE__);
  if (tokens.size() != 2 + paramb.num_types) {
    std::cout << "The first line of nep.txt should have " << paramb.num_types << " atom symbols."
              << std::endl;
    exit(1);
  }

  if (paramb.num_types == 1) {
    printf("Use the NEP%d potential with %d atom type.\n", paramb.version, paramb.num_types);
  } else {
    printf("Use the NEP%d potential with %d atom types.\n", paramb.version, paramb.num_types);
  }

  for (int n = 0; n < paramb.num_types; ++n) {
    int atomic_number = 0;
    for (int m = 0; m < NUM_ELEMENTS; ++m) {
      if (tokens[2 + n] == ELEMENTS[m]) {
        atomic_number = m + 1;
        break;
      }
    }
    paramb.atomic_numbers[n] = atomic_number - 1;
    printf("    type %d (%s with Z = %d).\n", n, tokens[2 + n].c_str(), atomic_number);
  }


  // cutoff 4.2 3.7 80 47 1
  tokens = get_tokens(input);
  if (tokens.size() != 5 && tokens.size() != 8) {
    std::cout << "This line should be cutoff rc_radial rc_angular MN_radial MN_angular "
                 "[radial_factor] [angular_factor] [zbl_factor].\n";
    exit(1);
  }
  paramb.rc_radial = get_float_from_token(tokens[1], __FILE__, __LINE__);
  paramb.rc_angular = get_float_from_token(tokens[2], __FILE__, __LINE__);
  printf("    radial cutoff = %g A.\n", paramb.rc_radial);
  printf("    angular cutoff = %g A.\n", paramb.rc_angular);

  int MN_radial = get_int_from_token(tokens[3], __FILE__, __LINE__);
  int MN_angular = get_int_from_token(tokens[4], __FILE__, __LINE__);
  printf("    MN_radial = %d.\n", MN_radial);
  if (MN_radial > 819) {
    std::cout << "The maximum number of neighbors exceeds 819. Please reduce this value."
              << std::endl;
    exit(1);
  }
  paramb.MN_radial = int(ceil(MN_radial * 1.25));
  paramb.MN_angular = int(ceil(MN_angular * 1.25));
  printf("    enlarged MN_radial = %d.\n", paramb.MN_radial);
  printf("    enlarged MN_angular = %d.\n", paramb.MN_angular);

  if (tokens.size() == 8) {
    paramb.typewise_cutoff_radial_factor = get_float_from_token(tokens[5], __FILE__, __LINE__);
    paramb.typewise_cutoff_angular_factor = get_float_from_token(tokens[6], __FILE__, __LINE__);
    paramb.typewise_cutoff_zbl_factor = get_float_from_token(tokens[7], __FILE__, __LINE__);
    if (paramb.typewise_cutoff_radial_factor > 0.0f) {
      paramb.use_typewise_cutoff = true;
    }
    if (paramb.typewise_cutoff_zbl_factor > 0.0f) {
      paramb.use_typewise_cutoff_zbl = true;
    }
  }
#ifdef USE_TABLE
  if (paramb.use_typewise_cutoff) {
    PRINT_INPUT_ERROR("Cannot use tabulated radial functions with typewise cutoff.");
  }
#endif

  // n_max 10 8
  tokens = get_tokens(input);
  if (tokens.size() != 3) {
    std::cout << "This line should be n_max n_max_radial n_max_angular." << std::endl;
    exit(1);
  }
  paramb.n_max_radial = get_int_from_token(tokens[1], __FILE__, __LINE__);
  paramb.n_max_angular = get_int_from_token(tokens[2], __FILE__, __LINE__);
  printf("    n_max_radial = %d.\n", paramb.n_max_radial);
  printf("    n_max_angular = %d.\n", paramb.n_max_angular);

  // basis_size 10 8
  tokens = get_tokens(input);
  if (tokens.size() != 3) {
    std::cout << "This line should be basis_size basis_size_radial basis_size_angular."
              << std::endl;
    exit(1);
  }
  paramb.basis_size_radial = get_int_from_token(tokens[1], __FILE__, __LINE__);
  paramb.basis_size_angular = get_int_from_token(tokens[2], __FILE__, __LINE__);
  printf("    basis_size_radial = %d.\n", paramb.basis_size_radial);
  printf("    basis_size_angular = %d.\n", paramb.basis_size_angular);

  // l_max
  tokens = get_tokens(input);
  if (tokens.size() != 4) {
    std::cout << "This line should be l_max l_max_3body l_max_4body l_max_5body." << std::endl;
    exit(1);
  }

  paramb.L_max = get_int_from_token(tokens[1], __FILE__, __LINE__);
  printf("    l_max_3body = %d.\n", paramb.L_max);
  paramb.num_L = paramb.L_max;

  int L_max_4body = get_int_from_token(tokens[2], __FILE__, __LINE__);
  int L_max_5body = get_int_from_token(tokens[3], __FILE__, __LINE__);
  printf("    l_max_4body = %d.\n", L_max_4body);
  printf("    l_max_5body = %d.\n", L_max_5body);
  if (L_max_4body == 2) {
    paramb.num_L += 1;
  }
  if (L_max_5body == 1) {
    paramb.num_L += 1;
  }

  paramb.dim_angular = (paramb.n_max_angular + 1) * paramb.num_L;

  // ANN
  tokens = get_tokens(input);
  if (tokens.size() != 3) {
    std::cout << "This line should be ANN num_neurons 0." << std::endl;
    exit(1);
  }
  annmb.num_neurons1 = get_int_from_token(tokens[1], __FILE__, __LINE__);
  annmb.dim = (paramb.n_max_radial + 1) + paramb.dim_angular;
  nep_model_type = paramb.model_type;
  if (paramb.model_type == 3) {
    annmb.dim += 1;
  }
  printf("    ANN = %d-%d-1.\n", annmb.dim, annmb.num_neurons1);

  // calculated parameters:
  rc = paramb.rc_radial; // largest cutoff
  paramb.rcinv_radial = 1.0f / paramb.rc_radial;
  paramb.rcinv_angular = 1.0f / paramb.rc_angular;
  paramb.num_types_sq = paramb.num_types * paramb.num_types;

  if (paramb.version == 3) {
    annmb.num_para_ann = (annmb.dim + 2) * annmb.num_neurons1 + 1;
  } else if (paramb.version == 4) {
    annmb.num_para_ann = (annmb.dim + 2) * annmb.num_neurons1 * paramb.num_types + 1;
  } else {
    annmb.num_para_ann = ((annmb.dim + 2) * annmb.num_neurons1 + 1) * paramb.num_types + 1;
  }
  if (paramb.model_type == 2) {
    // Polarizability models have twice as many parameters
    annmb.num_para_ann *= 2;
  }
  printf("    number of neural network parameters = %d.\n", annmb.num_para_ann);
  int num_para_descriptor =
    paramb.num_types_sq * ((paramb.n_max_radial + 1) * (paramb.basis_size_radial + 1) +
                           (paramb.n_max_angular + 1) * (paramb.basis_size_angular + 1));
  printf("    number of descriptor parameters = %d.\n", num_para_descriptor);
  annmb.num_para = annmb.num_para_ann + num_para_descriptor;
  printf("    total number of parameters = %d.\n", annmb.num_para);

  paramb.num_c_radial =
    paramb.num_types_sq * (paramb.n_max_radial + 1) * (paramb.basis_size_radial + 1);

  // NN and descriptor parameters
  std::vector<float> parameters(annmb.num_para);
  for (int n = 0; n < annmb.num_para; ++n) {
    tokens = get_tokens(input);
    parameters[n] = get_float_from_token(tokens[0], __FILE__, __LINE__);
  }
  nep_data.parameters.resize(annmb.num_para);
  nep_data.parameters.copy_from_host(parameters.data());
  update_potential(nep_data.parameters.data(), annmb);
  for (int d = 0; d < annmb.dim; ++d) {
    tokens = get_tokens(input);
    paramb.q_scaler[d] = get_float_from_token(tokens[0], __FILE__, __LINE__);
  }


  nep_data.f12x.resize(num_atoms * paramb.MN_angular);
  nep_data.f12y.resize(num_atoms * paramb.MN_angular);
  nep_data.f12z.resize(num_atoms * paramb.MN_angular);
  nep_data.NN_radial.resize(num_atoms);
  nep_data.NL_radial.resize(num_atoms * paramb.MN_radial);
  nep_data.NN_angular.resize(num_atoms);
  nep_data.NL_angular.resize(num_atoms * paramb.MN_angular);
  nep_data.Fp.resize(num_atoms * annmb.dim);
  nep_data.sum_fxyz.resize(num_atoms * (paramb.n_max_angular + 1) * NUM_OF_ABC);
  nep_data.cell_count.resize(num_atoms);
  nep_data.cell_count_sum.resize(num_atoms);
  nep_data.cell_contents.resize(num_atoms);
  nep_data.cpu_NN_radial.resize(num_atoms);
  nep_data.cpu_NN_angular.resize(num_atoms);

#ifdef USE_TABLE
  construct_table(parameters.data());
  printf("    use tabulated radial functions to speed up.\n");
#endif

}

ILP_NEP_GR_HBN::~ILP_NEP_GR_HBN(void)
{
  // nothing
}