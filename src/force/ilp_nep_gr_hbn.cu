#include "hip/hip_runtime.h"
/*
    Copyright 2017 Zheyong Fan and GPUMD development team
    This file is part of GPUMD.
    GPUMD is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUMD is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUMD.  If not, see <http://www.gnu.org/licenses/>.
*/

/*----------------------------------------------------------------------------80
The class dealing with the interlayer potential(ILP) and SW.
TODO:
------------------------------------------------------------------------------*/

#include "ilp_nep_gr_hbn.cuh"
#include "neighbor.cuh"
#include "utilities/error.cuh"
#include "utilities/common.cuh"
#include "utilities/gpu_macro.cuh"
#include "utilities/nep_utilities.cuh"
#include <fstream>
#include <iostream>
#include <string>
#include <vector>

#define BLOCK_SIZE_FORCE 128

// there are most 3 intra-layer neighbors for graphene and h-BN
#define NNEI 3

const std::string ELEMENTS[NUM_ELEMENTS] = {
  "H",  "He", "Li", "Be", "B",  "C",  "N",  "O",  "F",  "Ne", "Na", "Mg", "Al", "Si", "P",  "S",
  "Cl", "Ar", "K",  "Ca", "Sc", "Ti", "V",  "Cr", "Mn", "Fe", "Co", "Ni", "Cu", "Zn", "Ga", "Ge",
  "As", "Se", "Br", "Kr", "Rb", "Sr", "Y",  "Zr", "Nb", "Mo", "Tc", "Ru", "Rh", "Pd", "Ag", "Cd",
  "In", "Sn", "Sb", "Te", "I",  "Xe", "Cs", "Ba", "La", "Ce", "Pr", "Nd", "Pm", "Sm", "Eu", "Gd",
  "Tb", "Dy", "Ho", "Er", "Tm", "Yb", "Lu", "Hf", "Ta", "W",  "Re", "Os", "Ir", "Pt", "Au", "Hg",
  "Tl", "Pb", "Bi", "Po", "At", "Rn", "Fr", "Ra", "Ac", "Th", "Pa", "U",  "Np", "Pu"};


ILP_NEP_GR_HBN::ILP_NEP_GR_HBN(FILE* fid_ilp, const char* file_nep, int num_types, int num_atoms)
{
  // read ILP TMD potential parameter
  printf("Use %d-element ILP potential with elements:\n", num_types);
  if (!(num_types >= 1 && num_types <= MAX_TYPE_ILP_GR_HBN)) {
    PRINT_INPUT_ERROR("Incorrect type number of ILP_NEP_GR_HBN parameters.\n");
  }
  for (int n = 0; n < num_types; ++n) {
    char atom_symbol[10];
    int count = fscanf(fid_ilp, "%s", atom_symbol);
    PRINT_SCANF_ERROR(count, 1, "Reading error for ILP_NEP_GR_HBN potential.");
    printf(" %s", atom_symbol);
  }
  printf("\n");

  // read parameters
  float beta, alpha, delta, epsilon, C, d, sR;
  float reff, C6, S, rcut_ilp, rcut_global;
  rc = 0.0;
  for (int n = 0; n < num_types; ++n) {
    for (int m = 0; m < num_types; ++m) {
      int count = fscanf(fid_ilp, "%f%f%f%f%f%f%f%f%f%f%f%f", \
      &beta, &alpha, &delta, &epsilon, &C, &d, &sR, &reff, &C6, &S, \
      &rcut_ilp, &rcut_global);
      PRINT_SCANF_ERROR(count, 12, "Reading error for ILP_NEP_GR_HBN potential.");

      ilp_para.C[n][m] = C;
      ilp_para.C_6[n][m] = C6;
      ilp_para.d[n][m] = d;
      ilp_para.d_Seff[n][m] = d / sR / reff;
      ilp_para.epsilon[n][m] = epsilon;
      ilp_para.z0[n][m] = beta;
      ilp_para.lambda[n][m] = alpha / beta;
      ilp_para.delta2inv[n][m] = 1.0 / (delta * delta);
      ilp_para.S[n][m] = S;
      ilp_para.rcutsq_ilp[n][m] = rcut_ilp * rcut_ilp;
      ilp_para.rcut_global[n][m] = rcut_global;
      float meV = 1e-3 * S;
      ilp_para.C[n][m] *= meV;
      ilp_para.C_6[n][m] *= meV;
      ilp_para.epsilon[n][m] *= meV;

      if (rc < rcut_global)
        rc = rcut_global;
    }
  }


  // initialize neighbor lists and some temp vectors
  int max_neighbor_number = min(num_atoms, CUDA_MAX_NL_GR_HBN);
  ilp_data.NN.resize(num_atoms);
  ilp_data.NL.resize(num_atoms * max_neighbor_number);
  ilp_data.cell_count.resize(num_atoms);
  ilp_data.cell_count_sum.resize(num_atoms);
  ilp_data.cell_contents.resize(num_atoms);

  // init ilp neighbor list
  ilp_data.ilp_NN.resize(num_atoms);
  ilp_data.ilp_NL.resize(num_atoms * MAX_ILP_NEIGHBOR_GR_HBN);
  ilp_data.reduce_NL.resize(num_atoms * max_neighbor_number);
  ilp_data.big_ilp_NN.resize(num_atoms);
  ilp_data.big_ilp_NL.resize(num_atoms * MAX_BIG_ILP_NEIGHBOR_GR_HBN);

  ilp_data.f12x.resize(num_atoms * max_neighbor_number);
  ilp_data.f12y.resize(num_atoms * max_neighbor_number);
  ilp_data.f12z.resize(num_atoms * max_neighbor_number);

  ilp_data.f12x_ilp_neigh.resize(num_atoms * MAX_ILP_NEIGHBOR_GR_HBN);
  ilp_data.f12y_ilp_neigh.resize(num_atoms * MAX_ILP_NEIGHBOR_GR_HBN);
  ilp_data.f12z_ilp_neigh.resize(num_atoms * MAX_ILP_NEIGHBOR_GR_HBN);


  // init constant cutoff coeff
  float h_tap_coeff[8] = \
    {1.0f, 0.0f, 0.0f, 0.0f, -35.0f, 84.0f, -70.0f, 20.0f};
  CHECK(gpuMemcpyToSymbol(Tap_coeff_tmd, h_tap_coeff, 8 * sizeof(float)));

  // set ilp_flag to 1
  ilp_flag = 1;


  std::ifstream input(file_nep);
  if (!input.is_open()) {
    std::cout << "Failed to open " << file_nep << std::endl;
    exit(1);
  }

  // nep3 1 C
  std::vector<std::string> tokens = get_tokens(input);
  if (tokens.size() < 3) {
    std::cout << "The first line of nep.txt should have at least 3 items." << std::endl;
    exit(1);
  }
  if (tokens[0] == "nep3") {
    paramb.version = 3;
  } else if (tokens[0] == "nep4") {
    paramb.version = 4;
  } else if (tokens[0] == "nep5") {
    paramb.version = 5;
  } else {
    std::cout << tokens[0]
              << " is an unsupported NEP model. We only support NEP3 and NEP4 models now."
              << std::endl;
    exit(1);
  }
  paramb.num_types = get_int_from_token(tokens[1], __FILE__, __LINE__);
  if (tokens.size() != 2 + paramb.num_types) {
    std::cout << "The first line of nep.txt should have " << paramb.num_types << " atom symbols."
              << std::endl;
    exit(1);
  }

  if (paramb.num_types == 1) {
    printf("Use the NEP%d potential with %d atom type.\n", paramb.version, paramb.num_types);
  } else {
    printf("Use the NEP%d potential with %d atom types.\n", paramb.version, paramb.num_types);
  }

  for (int n = 0; n < paramb.num_types; ++n) {
    int atomic_number = 0;
    for (int m = 0; m < NUM_ELEMENTS; ++m) {
      if (tokens[2 + n] == ELEMENTS[m]) {
        atomic_number = m + 1;
        break;
      }
    }
    paramb.atomic_numbers[n] = atomic_number - 1;
    printf("    type %d (%s with Z = %d).\n", n, tokens[2 + n].c_str(), atomic_number);
  }


  // cutoff 4.2 3.7 80 47 1
  tokens = get_tokens(input);
  if (tokens.size() != 5 && tokens.size() != 8) {
    std::cout << "This line should be cutoff rc_radial rc_angular MN_radial MN_angular "
                 "[radial_factor] [angular_factor] [zbl_factor].\n";
    exit(1);
  }
  paramb.rc_radial = get_float_from_token(tokens[1], __FILE__, __LINE__);
  paramb.rc_angular = get_float_from_token(tokens[2], __FILE__, __LINE__);
  printf("    radial cutoff = %g A.\n", paramb.rc_radial);
  printf("    angular cutoff = %g A.\n", paramb.rc_angular);

  int MN_radial = get_int_from_token(tokens[3], __FILE__, __LINE__);
  int MN_angular = get_int_from_token(tokens[4], __FILE__, __LINE__);
  printf("    MN_radial = %d.\n", MN_radial);
  if (MN_radial > 819) {
    std::cout << "The maximum number of neighbors exceeds 819. Please reduce this value."
              << std::endl;
    exit(1);
  }
  paramb.MN_radial = int(ceil(MN_radial * 1.25));
  paramb.MN_angular = int(ceil(MN_angular * 1.25));
  printf("    enlarged MN_radial = %d.\n", paramb.MN_radial);
  printf("    enlarged MN_angular = %d.\n", paramb.MN_angular);

  if (tokens.size() == 8) {
    paramb.typewise_cutoff_radial_factor = get_float_from_token(tokens[5], __FILE__, __LINE__);
    paramb.typewise_cutoff_angular_factor = get_float_from_token(tokens[6], __FILE__, __LINE__);
    paramb.typewise_cutoff_zbl_factor = get_float_from_token(tokens[7], __FILE__, __LINE__);
    if (paramb.typewise_cutoff_radial_factor > 0.0f) {
      paramb.use_typewise_cutoff = true;
    }
    if (paramb.typewise_cutoff_zbl_factor > 0.0f) {
      paramb.use_typewise_cutoff_zbl = true;
    }
  }
#ifdef USE_TABLE
  if (paramb.use_typewise_cutoff) {
    PRINT_INPUT_ERROR("Cannot use tabulated radial functions with typewise cutoff.");
  }
#endif

  // n_max 10 8
  tokens = get_tokens(input);
  if (tokens.size() != 3) {
    std::cout << "This line should be n_max n_max_radial n_max_angular." << std::endl;
    exit(1);
  }
  paramb.n_max_radial = get_int_from_token(tokens[1], __FILE__, __LINE__);
  paramb.n_max_angular = get_int_from_token(tokens[2], __FILE__, __LINE__);
  printf("    n_max_radial = %d.\n", paramb.n_max_radial);
  printf("    n_max_angular = %d.\n", paramb.n_max_angular);

  // basis_size 10 8
  tokens = get_tokens(input);
  if (tokens.size() != 3) {
    std::cout << "This line should be basis_size basis_size_radial basis_size_angular."
              << std::endl;
    exit(1);
  }
  paramb.basis_size_radial = get_int_from_token(tokens[1], __FILE__, __LINE__);
  paramb.basis_size_angular = get_int_from_token(tokens[2], __FILE__, __LINE__);
  printf("    basis_size_radial = %d.\n", paramb.basis_size_radial);
  printf("    basis_size_angular = %d.\n", paramb.basis_size_angular);

  // l_max
  tokens = get_tokens(input);
  if (tokens.size() != 4) {
    std::cout << "This line should be l_max l_max_3body l_max_4body l_max_5body." << std::endl;
    exit(1);
  }

  paramb.L_max = get_int_from_token(tokens[1], __FILE__, __LINE__);
  printf("    l_max_3body = %d.\n", paramb.L_max);
  paramb.num_L = paramb.L_max;

  int L_max_4body = get_int_from_token(tokens[2], __FILE__, __LINE__);
  int L_max_5body = get_int_from_token(tokens[3], __FILE__, __LINE__);
  printf("    l_max_4body = %d.\n", L_max_4body);
  printf("    l_max_5body = %d.\n", L_max_5body);
  if (L_max_4body == 2) {
    paramb.num_L += 1;
  }
  if (L_max_5body == 1) {
    paramb.num_L += 1;
  }

  paramb.dim_angular = (paramb.n_max_angular + 1) * paramb.num_L;

  // ANN
  tokens = get_tokens(input);
  if (tokens.size() != 3) {
    std::cout << "This line should be ANN num_neurons 0." << std::endl;
    exit(1);
  }
  annmb.num_neurons1 = get_int_from_token(tokens[1], __FILE__, __LINE__);
  annmb.dim = (paramb.n_max_radial + 1) + paramb.dim_angular;
  nep_model_type = paramb.model_type;
  if (paramb.model_type == 3) {
    annmb.dim += 1;
  }
  printf("    ANN = %d-%d-1.\n", annmb.dim, annmb.num_neurons1);

  // calculated parameters:
  rc = paramb.rc_radial; // largest cutoff
  paramb.rcinv_radial = 1.0f / paramb.rc_radial;
  paramb.rcinv_angular = 1.0f / paramb.rc_angular;
  paramb.num_types_sq = paramb.num_types * paramb.num_types;

  if (paramb.version == 3) {
    annmb.num_para_ann = (annmb.dim + 2) * annmb.num_neurons1 + 1;
  } else if (paramb.version == 4) {
    annmb.num_para_ann = (annmb.dim + 2) * annmb.num_neurons1 * paramb.num_types + 1;
  } else {
    annmb.num_para_ann = ((annmb.dim + 2) * annmb.num_neurons1 + 1) * paramb.num_types + 1;
  }
  if (paramb.model_type == 2) {
    // Polarizability models have twice as many parameters
    annmb.num_para_ann *= 2;
  }
  printf("    number of neural network parameters = %d.\n", annmb.num_para_ann);
  int num_para_descriptor =
    paramb.num_types_sq * ((paramb.n_max_radial + 1) * (paramb.basis_size_radial + 1) +
                           (paramb.n_max_angular + 1) * (paramb.basis_size_angular + 1));
  printf("    number of descriptor parameters = %d.\n", num_para_descriptor);
  annmb.num_para = annmb.num_para_ann + num_para_descriptor;
  printf("    total number of parameters = %d.\n", annmb.num_para);

  paramb.num_c_radial =
    paramb.num_types_sq * (paramb.n_max_radial + 1) * (paramb.basis_size_radial + 1);

  // NN and descriptor parameters
  std::vector<float> parameters(annmb.num_para);
  for (int n = 0; n < annmb.num_para; ++n) {
    tokens = get_tokens(input);
    parameters[n] = get_float_from_token(tokens[0], __FILE__, __LINE__);
  }
  nep_data.parameters.resize(annmb.num_para);
  nep_data.parameters.copy_from_host(parameters.data());
  update_potential(nep_data.parameters.data(), annmb);
  for (int d = 0; d < annmb.dim; ++d) {
    tokens = get_tokens(input);
    paramb.q_scaler[d] = get_float_from_token(tokens[0], __FILE__, __LINE__);
  }


  nep_data.f12x.resize(num_atoms * paramb.MN_angular);
  nep_data.f12y.resize(num_atoms * paramb.MN_angular);
  nep_data.f12z.resize(num_atoms * paramb.MN_angular);
  nep_data.NN_radial.resize(num_atoms);
  nep_data.NL_radial.resize(num_atoms * paramb.MN_radial);
  nep_data.NN_angular.resize(num_atoms);
  nep_data.NL_angular.resize(num_atoms * paramb.MN_angular);
  nep_data.Fp.resize(num_atoms * annmb.dim);
  nep_data.sum_fxyz.resize(num_atoms * (paramb.n_max_angular + 1) * NUM_OF_ABC);
  nep_data.cell_count.resize(num_atoms);
  nep_data.cell_count_sum.resize(num_atoms);
  nep_data.cell_contents.resize(num_atoms);
  nep_data.cpu_NN_radial.resize(num_atoms);
  nep_data.cpu_NN_angular.resize(num_atoms);

#ifdef USE_TABLE
  construct_table(parameters.data());
  printf("    use tabulated radial functions to speed up.\n");
#endif

}

ILP_NEP_GR_HBN::~ILP_NEP_GR_HBN(void)
{
  // nothing
}

void ILP_NEP_GR_HBN::update_potential(float* parameters, ANN& ann)
{
  float* pointer = parameters;
  for (int t = 0; t < paramb.num_types; ++t) {
    if (t > 0 && paramb.version == 3) { // Use the same set of NN parameters for NEP3
      pointer -= (ann.dim + 2) * ann.num_neurons1;
    }
    ann.w0[t] = pointer;
    pointer += ann.num_neurons1 * ann.dim;
    ann.b0[t] = pointer;
    pointer += ann.num_neurons1;
    ann.w1[t] = pointer;
    pointer += ann.num_neurons1;
    if (paramb.version == 5) {
      pointer += 1; // one extra bias for NEP5 stored in ann.w1[t]
    }
  }
  ann.b1 = pointer;
  pointer += 1;


  ann.c = pointer;
}

static __global__ void find_neighbor_list_nep(
  NEP::ParaMB paramb,
  const int N,
  const int N1,
  const int N2,
  const int nx,
  const int ny,
  const int nz,
  const Box box,
  const int* g_type,
  const int* __restrict__ g_cell_count,
  const int* __restrict__ g_cell_count_sum,
  const int* __restrict__ g_cell_contents,
  const double* __restrict__ g_x,
  const double* __restrict__ g_y,
  const double* __restrict__ g_z,
  int* g_NN_radial,
  int* g_NL_radial,
  int* g_NN_angular,
  int* g_NL_angular)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1;
  if (n1 >= N2) {
    return;
  }

  double x1 = g_x[n1];
  double y1 = g_y[n1];
  double z1 = g_z[n1];
  int t1 = g_type[n1];
  int count_radial = 0;
  int count_angular = 0;

  int cell_id;
  int cell_id_x;
  int cell_id_y;
  int cell_id_z;
  find_cell_id(
    box,
    x1,
    y1,
    z1,
    2.0f * paramb.rcinv_radial,
    nx,
    ny,
    nz,
    cell_id_x,
    cell_id_y,
    cell_id_z,
    cell_id);

  const int z_lim = box.pbc_z ? 2 : 0;
  const int y_lim = box.pbc_y ? 2 : 0;
  const int x_lim = box.pbc_x ? 2 : 0;

  for (int zz = -z_lim; zz <= z_lim; ++zz) {
    for (int yy = -y_lim; yy <= y_lim; ++yy) {
      for (int xx = -x_lim; xx <= x_lim; ++xx) {
        int neighbor_cell = cell_id + zz * nx * ny + yy * nx + xx;
        if (cell_id_x + xx < 0)
          neighbor_cell += nx;
        if (cell_id_x + xx >= nx)
          neighbor_cell -= nx;
        if (cell_id_y + yy < 0)
          neighbor_cell += ny * nx;
        if (cell_id_y + yy >= ny)
          neighbor_cell -= ny * nx;
        if (cell_id_z + zz < 0)
          neighbor_cell += nz * ny * nx;
        if (cell_id_z + zz >= nz)
          neighbor_cell -= nz * ny * nx;

        const int num_atoms_neighbor_cell = g_cell_count[neighbor_cell];
        const int num_atoms_previous_cells = g_cell_count_sum[neighbor_cell];

        for (int m = 0; m < num_atoms_neighbor_cell; ++m) {
          const int n2 = g_cell_contents[num_atoms_previous_cells + m];

          if (n2 < N1 || n2 >= N2 || n1 == n2) {
            continue;
          }

          double x12double = g_x[n2] - x1;
          double y12double = g_y[n2] - y1;
          double z12double = g_z[n2] - z1;
          apply_mic(box, x12double, y12double, z12double);
          float x12 = float(x12double), y12 = float(y12double), z12 = float(z12double);
          float d12_square = x12 * x12 + y12 * y12 + z12 * z12;

          int t2 = g_type[n2];
          float rc_radial = paramb.rc_radial;
          float rc_angular = paramb.rc_angular;
          if (paramb.use_typewise_cutoff) {
            int z1 = paramb.atomic_numbers[t1];
            int z2 = paramb.atomic_numbers[t2];
            rc_radial = min(
              (COVALENT_RADIUS[z1] + COVALENT_RADIUS[z2]) * paramb.typewise_cutoff_radial_factor,
              rc_radial);
            rc_angular = min(
              (COVALENT_RADIUS[z1] + COVALENT_RADIUS[z2]) * paramb.typewise_cutoff_angular_factor,
              rc_angular);
          }

          if (d12_square >= rc_radial * rc_radial) {
            continue;
          }

          g_NL_radial[count_radial++ * N + n1] = n2;

          if (d12_square < rc_angular * rc_angular) {
            g_NL_angular[count_angular++ * N + n1] = n2;
          }
        }
      }
    }
  }

  g_NN_radial[n1] = count_radial;
  g_NN_angular[n1] = count_angular;
}

static __global__ void find_descriptor(
  NEP::ParaMB paramb,
  NEP::ANN annmb,
  const int N,
  const int N1,
  const int N2,
  const Box box,
  const int* g_NN,
  const int* g_NL,
  const int* g_NN_angular,
  const int* g_NL_angular,
  const int* __restrict__ g_type,
  const double* __restrict__ g_x,
  const double* __restrict__ g_y,
  const double* __restrict__ g_z,
#ifdef USE_TABLE
  const float* __restrict__ g_gn_radial,
  const float* __restrict__ g_gn_angular,
#endif
  double* g_pe,
  float* g_Fp,
  double* g_virial,
  float* g_sum_fxyz)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1;
  if (n1 < N2) {
    int t1 = g_type[n1];
    double x1 = g_x[n1];
    double y1 = g_y[n1];
    double z1 = g_z[n1];
    float q[MAX_DIM] = {0.0f};

    // get radial descriptors
    for (int i1 = 0; i1 < g_NN[n1]; ++i1) {
      int n2 = g_NL[n1 + N * i1];
      double x12double = g_x[n2] - x1;
      double y12double = g_y[n2] - y1;
      double z12double = g_z[n2] - z1;
      apply_mic(box, x12double, y12double, z12double);
      float x12 = float(x12double), y12 = float(y12double), z12 = float(z12double);
      float d12 = sqrt(x12 * x12 + y12 * y12 + z12 * z12);

#ifdef USE_TABLE
      int index_left, index_right;
      float weight_left, weight_right;
      find_index_and_weight(
        d12 * paramb.rcinv_radial, index_left, index_right, weight_left, weight_right);
      int t12 = t1 * paramb.num_types + g_type[n2];
      for (int n = 0; n <= paramb.n_max_radial; ++n) {
        q[n] +=
          g_gn_radial[(index_left * paramb.num_types_sq + t12) * (paramb.n_max_radial + 1) + n] *
            weight_left +
          g_gn_radial[(index_right * paramb.num_types_sq + t12) * (paramb.n_max_radial + 1) + n] *
            weight_right;
      }
#else
      float fc12;
      int t2 = g_type[n2];
      float rc = paramb.rc_radial;
      if (paramb.use_typewise_cutoff) {
        rc = min(
          (COVALENT_RADIUS[paramb.atomic_numbers[t1]] +
           COVALENT_RADIUS[paramb.atomic_numbers[t2]]) *
            paramb.typewise_cutoff_radial_factor,
          rc);
      }
      float rcinv = 1.0f / rc;
      find_fc(rc, rcinv, d12, fc12);
      float fn12[MAX_NUM_N];

      find_fn(paramb.basis_size_radial, rcinv, d12, fc12, fn12);
      for (int n = 0; n <= paramb.n_max_radial; ++n) {
        float gn12 = 0.0f;
        for (int k = 0; k <= paramb.basis_size_radial; ++k) {
          int c_index = (n * (paramb.basis_size_radial + 1) + k) * paramb.num_types_sq;
          c_index += t1 * paramb.num_types + t2;
          gn12 += fn12[k] * annmb.c[c_index];
        }
        q[n] += gn12;
      }
#endif
    }

    // get angular descriptors
    for (int n = 0; n <= paramb.n_max_angular; ++n) {
      float s[NUM_OF_ABC] = {0.0f};
      for (int i1 = 0; i1 < g_NN_angular[n1]; ++i1) {
        int n2 = g_NL_angular[n1 + N * i1];
        double x12double = g_x[n2] - x1;
        double y12double = g_y[n2] - y1;
        double z12double = g_z[n2] - z1;
        apply_mic(box, x12double, y12double, z12double);
        float x12 = float(x12double), y12 = float(y12double), z12 = float(z12double);
        float d12 = sqrt(x12 * x12 + y12 * y12 + z12 * z12);
#ifdef USE_TABLE
        int index_left, index_right;
        float weight_left, weight_right;
        find_index_and_weight(
          d12 * paramb.rcinv_angular, index_left, index_right, weight_left, weight_right);
        int t12 = t1 * paramb.num_types + g_type[n2];
        float gn12 =
          g_gn_angular[(index_left * paramb.num_types_sq + t12) * (paramb.n_max_angular + 1) + n] *
            weight_left +
          g_gn_angular[(index_right * paramb.num_types_sq + t12) * (paramb.n_max_angular + 1) + n] *
            weight_right;
        accumulate_s(paramb.L_max, d12, x12, y12, z12, gn12, s);
#else
        float fc12;
        int t2 = g_type[n2];
        float rc = paramb.rc_angular;
        if (paramb.use_typewise_cutoff) {
          rc = min(
            (COVALENT_RADIUS[paramb.atomic_numbers[t1]] +
             COVALENT_RADIUS[paramb.atomic_numbers[t2]]) *
              paramb.typewise_cutoff_angular_factor,
            rc);
        }
        float rcinv = 1.0f / rc;
        find_fc(rc, rcinv, d12, fc12);
        float fn12[MAX_NUM_N];
        find_fn(paramb.basis_size_angular, rcinv, d12, fc12, fn12);
        float gn12 = 0.0f;
        for (int k = 0; k <= paramb.basis_size_angular; ++k) {
          int c_index = (n * (paramb.basis_size_angular + 1) + k) * paramb.num_types_sq;
          c_index += t1 * paramb.num_types + t2 + paramb.num_c_radial;
          gn12 += fn12[k] * annmb.c[c_index];
        }
        accumulate_s(paramb.L_max, d12, x12, y12, z12, gn12, s);
#endif
      }
      find_q(paramb.L_max, paramb.num_L, paramb.n_max_angular + 1, n, s, q + (paramb.n_max_radial + 1));
      for (int abc = 0; abc < NUM_OF_ABC; ++abc) {
        g_sum_fxyz[(n * NUM_OF_ABC + abc) * N + n1] = s[abc];
      }
    }

    // nomalize descriptor
    for (int d = 0; d < annmb.dim; ++d) {
      q[d] = q[d] * paramb.q_scaler[d];
    }

    // get energy and energy gradient
    float F = 0.0f, Fp[MAX_DIM] = {0.0f};


    if (paramb.version == 5) {
      apply_ann_one_layer_nep5(
        annmb.dim,
        annmb.num_neurons1,
        annmb.w0[t1],
        annmb.b0[t1],
        annmb.w1[t1],
        annmb.b1,
        q,
        F,
        Fp);
    } else {
      apply_ann_one_layer(
        annmb.dim,
        annmb.num_neurons1,
        annmb.w0[t1],
        annmb.b0[t1],
        annmb.w1[t1],
        annmb.b1,
        q,
        F,
        Fp);
    }
    g_pe[n1] += F;

    for (int d = 0; d < annmb.dim; ++d) {
      g_Fp[d * N + n1] = Fp[d] * paramb.q_scaler[d];
    }
  }
}

static __global__ void find_force_radial(
  NEP::ParaMB paramb,
  NEP::ANN annmb,
  const int N,
  const int N1,
  const int N2,
  const Box box,
  const int* g_NN,
  const int* g_NL,
  const int* __restrict__ g_type,
  const double* __restrict__ g_x,
  const double* __restrict__ g_y,
  const double* __restrict__ g_z,
  const float* __restrict__ g_Fp,
#ifdef USE_TABLE
  const float* __restrict__ g_gnp_radial,
#endif
  double* g_fx,
  double* g_fy,
  double* g_fz,
  double* g_virial)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1;
  if (n1 < N2) {
    int t1 = g_type[n1];
    float s_fx = 0.0f;
    float s_fy = 0.0f;
    float s_fz = 0.0f;
    float s_sxx = 0.0f;
    float s_sxy = 0.0f;
    float s_sxz = 0.0f;
    float s_syx = 0.0f;
    float s_syy = 0.0f;
    float s_syz = 0.0f;
    float s_szx = 0.0f;
    float s_szy = 0.0f;
    float s_szz = 0.0f;
    double x1 = g_x[n1];
    double y1 = g_y[n1];
    double z1 = g_z[n1];
    for (int i1 = 0; i1 < g_NN[n1]; ++i1) {
      int n2 = g_NL[n1 + N * i1];
      int t2 = g_type[n2];
      double x12double = g_x[n2] - x1;
      double y12double = g_y[n2] - y1;
      double z12double = g_z[n2] - z1;
      apply_mic(box, x12double, y12double, z12double);
      float r12[3] = {float(x12double), float(y12double), float(z12double)};
      float d12 = sqrt(r12[0] * r12[0] + r12[1] * r12[1] + r12[2] * r12[2]);
      float d12inv = 1.0f / d12;
      float f12[3] = {0.0f};
      float f21[3] = {0.0f};
#ifdef USE_TABLE
      int index_left, index_right;
      float weight_left, weight_right;
      find_index_and_weight(
        d12 * paramb.rcinv_radial, index_left, index_right, weight_left, weight_right);
      int t12 = t1 * paramb.num_types + t2;
      int t21 = t2 * paramb.num_types + t1;
      for (int n = 0; n <= paramb.n_max_radial; ++n) {
        float gnp12 =
          g_gnp_radial[(index_left * paramb.num_types_sq + t12) * (paramb.n_max_radial + 1) + n] *
            weight_left +
          g_gnp_radial[(index_right * paramb.num_types_sq + t12) * (paramb.n_max_radial + 1) + n] *
            weight_right;
        float gnp21 =
          g_gnp_radial[(index_left * paramb.num_types_sq + t21) * (paramb.n_max_radial + 1) + n] *
            weight_left +
          g_gnp_radial[(index_right * paramb.num_types_sq + t21) * (paramb.n_max_radial + 1) + n] *
            weight_right;
        float tmp12 = g_Fp[n1 + n * N] * gnp12 * d12inv;
        float tmp21 = g_Fp[n2 + n * N] * gnp21 * d12inv;
        for (int d = 0; d < 3; ++d) {
          f12[d] += tmp12 * r12[d];
          f21[d] -= tmp21 * r12[d];
        }
      }
#else
      float fc12, fcp12;
      float rc = paramb.rc_radial;
      if (paramb.use_typewise_cutoff) {
        rc = min(
          (COVALENT_RADIUS[paramb.atomic_numbers[t1]] +
           COVALENT_RADIUS[paramb.atomic_numbers[t2]]) *
            paramb.typewise_cutoff_radial_factor,
          rc);
      }
      float rcinv = 1.0f / rc;
      find_fc_and_fcp(rc, rcinv, d12, fc12, fcp12);
      float fn12[MAX_NUM_N];
      float fnp12[MAX_NUM_N];
      find_fn_and_fnp(paramb.basis_size_radial, rcinv, d12, fc12, fcp12, fn12, fnp12);
      for (int n = 0; n <= paramb.n_max_radial; ++n) {
        float gnp12 = 0.0f;
        float gnp21 = 0.0f;
        for (int k = 0; k <= paramb.basis_size_radial; ++k) {
          int c_index = (n * (paramb.basis_size_radial + 1) + k) * paramb.num_types_sq;
          gnp12 += fnp12[k] * annmb.c[c_index + t1 * paramb.num_types + t2];
          gnp21 += fnp12[k] * annmb.c[c_index + t2 * paramb.num_types + t1];
        }
        float tmp12 = g_Fp[n1 + n * N] * gnp12 * d12inv;
        float tmp21 = g_Fp[n2 + n * N] * gnp21 * d12inv;
        for (int d = 0; d < 3; ++d) {
          f12[d] += tmp12 * r12[d];
          f21[d] -= tmp21 * r12[d];
        }
      }
#endif
      s_fx += f12[0] - f21[0];
      s_fy += f12[1] - f21[1];
      s_fz += f12[2] - f21[2];

      s_sxx += r12[0] * f21[0];
      s_syy += r12[1] * f21[1];
      s_szz += r12[2] * f21[2];
      s_sxy += r12[0] * f21[1];
      s_sxz += r12[0] * f21[2];
      s_syx += r12[1] * f21[0];
      s_syz += r12[1] * f21[2];
      s_szx += r12[2] * f21[0];
      s_szy += r12[2] * f21[1];
    }
    g_fx[n1] += s_fx;
    g_fy[n1] += s_fy;
    g_fz[n1] += s_fz;
    // save virial
    // xx xy xz    0 3 4
    // yx yy yz    6 1 5
    // zx zy zz    7 8 2
    g_virial[n1 + 0 * N] += s_sxx;
    g_virial[n1 + 1 * N] += s_syy;
    g_virial[n1 + 2 * N] += s_szz;
    g_virial[n1 + 3 * N] += s_sxy;
    g_virial[n1 + 4 * N] += s_sxz;
    g_virial[n1 + 5 * N] += s_syz;
    g_virial[n1 + 6 * N] += s_syx;
    g_virial[n1 + 7 * N] += s_szx;
    g_virial[n1 + 8 * N] += s_szy;
  }
}

static __global__ void find_partial_force_angular(
  NEP::ParaMB paramb,
  NEP::ANN annmb,
  const int N,
  const int N1,
  const int N2,
  const Box box,
  const int* g_NN_angular,
  const int* g_NL_angular,
  const int* __restrict__ g_type,
  const double* __restrict__ g_x,
  const double* __restrict__ g_y,
  const double* __restrict__ g_z,
  const float* __restrict__ g_Fp,
  const float* __restrict__ g_sum_fxyz,
#ifdef USE_TABLE
  const float* __restrict__ g_gn_angular,
  const float* __restrict__ g_gnp_angular,
#endif
  float* g_f12x,
  float* g_f12y,
  float* g_f12z)
{
  int n1 = blockIdx.x * blockDim.x + threadIdx.x + N1;
  if (n1 < N2) {

    float Fp[MAX_DIM_ANGULAR] = {0.0f};
    float sum_fxyz[NUM_OF_ABC * MAX_NUM_N];
    for (int d = 0; d < paramb.dim_angular; ++d) {
      Fp[d] = g_Fp[(paramb.n_max_radial + 1 + d) * N + n1];
    }
    for (int d = 0; d < (paramb.n_max_angular + 1) * NUM_OF_ABC; ++d) {
      sum_fxyz[d] = g_sum_fxyz[d * N + n1];
    }

    int t1 = g_type[n1];
    double x1 = g_x[n1];
    double y1 = g_y[n1];
    double z1 = g_z[n1];
    for (int i1 = 0; i1 < g_NN_angular[n1]; ++i1) {
      int index = i1 * N + n1;
      int n2 = g_NL_angular[n1 + N * i1];
      double x12double = g_x[n2] - x1;
      double y12double = g_y[n2] - y1;
      double z12double = g_z[n2] - z1;
      apply_mic(box, x12double, y12double, z12double);
      float r12[3] = {float(x12double), float(y12double), float(z12double)};
      float d12 = sqrt(r12[0] * r12[0] + r12[1] * r12[1] + r12[2] * r12[2]);
      float f12[3] = {0.0f};
#ifdef USE_TABLE
      int index_left, index_right;
      float weight_left, weight_right;
      find_index_and_weight(
        d12 * paramb.rcinv_angular, index_left, index_right, weight_left, weight_right);
      int t12 = t1 * paramb.num_types + g_type[n2];
      for (int n = 0; n <= paramb.n_max_angular; ++n) {
        int index_left_all =
          (index_left * paramb.num_types_sq + t12) * (paramb.n_max_angular + 1) + n;
        int index_right_all =
          (index_right * paramb.num_types_sq + t12) * (paramb.n_max_angular + 1) + n;
        float gn12 =
          g_gn_angular[index_left_all] * weight_left + g_gn_angular[index_right_all] * weight_right;
        float gnp12 = g_gnp_angular[index_left_all] * weight_left +
                      g_gnp_angular[index_right_all] * weight_right;
        accumulate_f12(paramb.L_max, paramb.num_L, n, paramb.n_max_angular + 1, d12, r12, gn12, gnp12, Fp, sum_fxyz, f12);
      }
#else
      float fc12, fcp12;
      int t2 = g_type[n2];
      float rc = paramb.rc_angular;
      if (paramb.use_typewise_cutoff) {
        rc = min(
          (COVALENT_RADIUS[paramb.atomic_numbers[t1]] +
           COVALENT_RADIUS[paramb.atomic_numbers[t2]]) *
            paramb.typewise_cutoff_angular_factor,
          rc);
      }
      float rcinv = 1.0f / rc;
      find_fc_and_fcp(rc, rcinv, d12, fc12, fcp12);

      float fn12[MAX_NUM_N];
      float fnp12[MAX_NUM_N];
      find_fn_and_fnp(paramb.basis_size_angular, rcinv, d12, fc12, fcp12, fn12, fnp12);
      for (int n = 0; n <= paramb.n_max_angular; ++n) {
        float gn12 = 0.0f;
        float gnp12 = 0.0f;
        for (int k = 0; k <= paramb.basis_size_angular; ++k) {
          int c_index = (n * (paramb.basis_size_angular + 1) + k) * paramb.num_types_sq;
          c_index += t1 * paramb.num_types + t2 + paramb.num_c_radial;
          gn12 += fn12[k] * annmb.c[c_index];
          gnp12 += fnp12[k] * annmb.c[c_index];
        }
        accumulate_f12(paramb.L_max, paramb.num_L, n, paramb.n_max_angular + 1, d12, r12, gn12, gnp12, Fp, sum_fxyz, f12);
      }
#endif
      g_f12x[index] = f12[0];
      g_f12y[index] = f12[1];
      g_f12z[index] = f12[2];
    }
  }
}